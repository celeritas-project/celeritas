#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020-2022 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file RDemoKernel.cu
//---------------------------------------------------------------------------//
#include "RDemoKernel.hh"

#include <cmath>

#include "base/Assert.hh"
#include "base/KernelParamCalculator.device.hh"
#include "geometry/GeoTrackView.hh"

#include "ImageTrackView.hh"

using namespace celeritas;
using namespace demo_rasterizer;

namespace demo_rasterizer
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__device__ int geo_id(const GeoTrackView& geo)
{
    if (geo.is_outside())
        return -1;
    return geo.volume_id().get();
}

__global__ void trace_kernel(const GeoParamsCRefDevice geo_params,
                             const GeoStateRefDevice   geo_state,
                             const ImageData           image_state)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= image_state.dims[0])
        return;

    ImageTrackView image(image_state, tid);
    GeoTrackView   geo(geo_params, geo_state, tid);

    // Start track at the leftmost point in the requested direction
    geo = GeoTrackInitializer{image.start_pos(), image.start_dir()};

    int cur_id = geo_id(geo);

    // Track along each pixel
    for (unsigned int i = 0; i < image_state.dims[1]; ++i)
    {
        real_type pix_dist      = image_state.pixel_width;
        real_type max_dist      = 0;
        int       max_id        = cur_id;
        int       abort_counter = 32; // max number of crossings per pixel

        auto next = geo.find_next_step(pix_dist);
        while (next.boundary && pix_dist > 0)
        {
            CELER_ASSERT(next.distance <= pix_dist);
            // Move to geometry boundary
            pix_dist -= next.distance;

            if (max_id == cur_id)
            {
                max_dist += next.distance;
            }
            else if (next.distance > max_dist)
            {
                max_dist = next.distance;
                max_id   = cur_id;
            }

            // Cross surface and update post-crossing ID
            geo.move_to_boundary();
            geo.cross_boundary();
            cur_id = geo_id(geo);

            if (--abort_counter == 0)
            {
                // Reinitialize at end of pixel
                Real3 new_pos = image.start_pos();
                celeritas::axpy((i + 1) * image_state.pixel_width,
                                image.start_dir(),
                                &new_pos);
                geo      = GeoTrackInitializer{new_pos, image.start_dir()};
                pix_dist = 0;
            }
            if (pix_dist > 0)
            {
                // Next movement is to end of geo or pixel
                next = geo.find_next_step(pix_dist);
            }
        }

        if (pix_dist > 0)
        {
            // Move to pixel boundary
            geo.move_internal(pix_dist);
            if (pix_dist > max_dist)
            {
                max_dist = pix_dist;
                max_id   = cur_id;
            }
        }
        image.set_pixel(i, max_id);
    }
}
} // namespace

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
void trace(const GeoParamsCRefDevice& geo_params,
           const GeoStateRefDevice&   geo_state,
           const ImageData&           image)
{
    CELER_EXPECT(image);

    static const KernelParamCalculator calc_kernel_params(trace_kernel,
                                                          "trace");

    auto params = calc_kernel_params(image.dims[0]);
    trace_kernel<<<params.grid_size, params.block_size>>>(
        geo_params, geo_state, image);
    CELER_DEVICE_CHECK_ERROR();

    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());
}

//---------------------------------------------------------------------------//
} // namespace demo_rasterizer
