#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file RDemoKernel.cu
//---------------------------------------------------------------------------//
#include "RDemoKernel.hh"

#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "geometry/GeoTrackView.hh"
#include "ImageTrackView.hh"
#include <cmath>

using namespace celeritas;
using namespace demo_rasterizer;

namespace demo_rasterizer
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__device__ int geo_id(const GeoTrackView& geo)
{
    if (geo.is_outside())
        return -1;
    return geo.volume_id().get();
}

__global__ void trace_kernel(const GeoParamsPointers geo_params,
                             const GeoStatePointers  geo_state,
                             const ImagePointers     image_state)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= image_state.dims[0])
        return;

    ImageTrackView image(image_state, tid);
    GeoTrackView   geo(geo_params, geo_state, tid);

    // Start track at the leftmost point in the requested direction
    geo = GeoStateInitializer{image.start_pos(), image.start_dir()};

    int       cur_id   = geo_id(geo);
    real_type geo_dist = std::fmin(
        geo.next_step(), image_state.dims[1] * image_state.pixel_width);

    // Track along each pixel
    for (unsigned int i = 0; i < image_state.dims[1]; ++i)
    {
        real_type pix_dist = image_state.pixel_width;
        real_type max_dist = 0;
        int       max_id   = cur_id;
        while (geo_dist <= pix_dist)
        {
            // Move to geometry boundary
            pix_dist -= geo_dist;

            if (max_id == cur_id)
            {
                max_dist += geo_dist;
            }
            else if (geo_dist > max_dist)
            {
                max_dist = geo_dist;
                max_id   = cur_id;
            }

            // Cross surface
            geo.move_next_step();
            cur_id   = geo_id(geo);
            geo_dist = std::fmin(geo.next_step(),
                                 image_state.dims[1] * image_state.pixel_width);
        }

        // Move to pixel boundary
        geo_dist -= pix_dist;
        if (pix_dist > max_dist)
        {
            max_dist = pix_dist;
            max_id   = cur_id;
        }
        image.set_pixel(i, max_id);
    }
}
} // namespace

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
void trace(const GeoParamsPointers& geo_params,
           const GeoStatePointers&  geo_state,
           const ImagePointers&     image)
{
    CELER_EXPECT(image);

    static const KernelParamCalculator calc_kernel_params(trace_kernel,
                                                          "trace");

    auto params = calc_kernel_params(image.dims[0]);
    trace_kernel<<<params.grid_size, params.block_size>>>(
        geo_params, geo_state, image);
    CELER_CUDA_CHECK_ERROR();

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
} // namespace demo_rasterizer
