#include "hip/hip_runtime.h"
//---------------------------------*-Cudac-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file GCheckKernel.cu
//---------------------------------------------------------------------------//
#include "corecel/Assert.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "GCheckKernel.hh"
#include "celeritas/geo/GeoData.hh"
#include "celeritas/geo/GeoTrackView.hh"
#include "celeritas/field/LinearPropagator.hh"
#include "corecel/data/CollectionStateStore.hh"

using namespace celeritas;
using thrust::raw_pointer_cast;

namespace geo_check
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__device__ int geo_physid(const GeoTrackView& geo)
{
    if (geo.is_outside())
        return -1;
    return geo.volume_physid().get();
}

__global__ void gcheck_kernel(const GeoParamsCRefDevice  params,
                              const GeoStateRefDevice    state,
                              const GeoTrackInitializer* init,
                              int                        max_steps,
                              int*                       ids,
                              double*                    distances)
{
    CELER_EXPECT(params && state);
    CELER_EXPECT(max_steps > 0);

    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= state.size())
        return;

    celeritas::GeoTrackView     geo(params, state, tid);
    celeritas::LinearPropagator propagate(&geo);

    // Start track at the leftmost point in the requested direction
    geo = init[tid.get()];

    // Track along detector
    int istep = 0;
    do
    {
        // Propagate Save next-volume ID and distance to travel
        auto step        = propagate();
        if (step.boundary) geo.cross_boundary();
        ids[istep]       = geo.volume_physid().get();
        distances[istep] = step.distance;
        ++istep;
    } while (!geo.is_outside() && istep < max_steps);
}

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 *  Run tracking on the GPU
 */
GCheckOutput run_gpu(GCheckInput input)
{
    CELER_EXPECT(input.params);
    CELER_EXPECT(input.state);
    CELER_EXPECT(input.max_steps > 0);

    // Temporary device data for kernel
    thrust::device_vector<GeoTrackInitializer> tracks(input.init.begin(),
                                                      input.init.end());
    thrust::device_vector<int>    ids(input.init.size() * input.max_steps, -1);
    thrust::device_vector<double> distances(ids.size(), -1.0);

    // static const KernelParamCalculator calc_kernel_params(gcheck_kernel,
    // 							  "run_gpu");
    // auto launch_params = calc_launch_params(init.size());

    gcheck_kernel<<<1, 1>>>(input.params,
                            input.state,
                            raw_pointer_cast(tracks.data()),
                            input.max_steps,
                            raw_pointer_cast(ids.data()),
                            raw_pointer_cast(distances.data()));

    CELER_DEVICE_CHECK_ERROR();
    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Copy result back to CPU
    GCheckOutput result;

    // figure out how many valid steps returned
    size_type    nstep = 0;
    for (auto id : thrust::host_vector<int>(ids))
    {
        if (id < 0) break;
        ++nstep;
    }
    // Return exact vector size for proper comparison with CPU
    result.ids.resize(nstep);
    thrust::copy(ids.begin(), ids.begin() + nstep, result.ids.begin());

    result.distances.resize(nstep);
    thrust::copy(
        distances.begin(), distances.begin() + nstep, result.distances.begin());

    return result;
}

//---------------------------------------------------------------------------//
} // namespace geo_check
