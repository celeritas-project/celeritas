#include "hip/hip_runtime.h"
//---------------------------------*-Cudac-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file GCheckKernel.cu
//---------------------------------------------------------------------------//
#include "GCheckKernel.hh"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "corecel/Assert.hh"
#include "corecel/data/CollectionStateStore.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/field/LinearPropagator.hh"
#include "celeritas/geo/GeoData.hh"
#include "celeritas/geo/GeoTrackView.hh"

using namespace celeritas;
using thrust::raw_pointer_cast;

namespace geo_check
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void gcheck_kernel(const GeoParamsCRefDevice params,
                              const GeoStateRefDevice state,
                              GeoTrackInitializer const* init,
                              int max_steps,
                              int* ids,
                              double* distances)
{
    CELER_EXPECT(params && state);
    CELER_EXPECT(max_steps > 0);

    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= state.size())
        return;

    celeritas::GeoTrackView geo(params, state, tid);
    celeritas::LinearPropagator propagate(&geo);

    // Start track at the leftmost point in the requested direction
    geo = init[tid.get()];

    // Track along detector
    int istep = 0;
    do
    {
        // Propagate Save next-volume ID and distance to travel
        auto step = propagate();
        if (step.boundary)
            geo.cross_boundary();
        ids[istep] = physid(geo);
        distances[istep] = step.distance;
        ++istep;
    } while (!geo.is_outside() && istep < max_steps);
}

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 *  Run tracking on the GPU
 */
GCheckOutput run_gpu(GCheckInput input)
{
    CELER_EXPECT(input.params);
    CELER_EXPECT(input.state);
    CELER_EXPECT(input.max_steps > 0);

    // Temporary device data for kernel
    thrust::device_vector<GeoTrackInitializer> tracks(input.init.begin(),
                                                      input.init.end());
    thrust::device_vector<int> ids(input.init.size() * input.max_steps, -1);
    thrust::device_vector<double> distances(ids.size(), -1.0);

    gcheck_kernel<<<1, 1>>>(input.params,
                            input.state,
                            raw_pointer_cast(tracks.data()),
                            input.max_steps,
                            raw_pointer_cast(ids.data()),
                            raw_pointer_cast(distances.data()));

    CELER_DEVICE_CHECK_ERROR();
    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Copy result back to CPU
    GCheckOutput result;
    result.ids.resize(input.max_steps);
    thrust::copy(ids.begin(), ids.end(), result.ids.begin());

    // find exact vector size for proper comparison with CPU
    auto end_ids = std::find_if(
        result.ids.begin(), result.ids.end(), [](int id) { return id < 0; });
    result.ids.erase(end_ids, result.ids.end());

    size_type nstep = end_ids - result.ids.begin();
    result.distances.resize(nstep);
    thrust::copy(
        distances.begin(), distances.begin() + nstep, result.distances.begin());

    return result;
}

//---------------------------------------------------------------------------//
}  // namespace geo_check
