#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file KNDemoKernel.cu
//---------------------------------------------------------------------------//
#include "KNDemoKernel.hh"

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include "base/ArrayUtils.hh"
#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/SecondaryAllocatorView.hh"
#include "physics/em/detail/KleinNishinaInteractor.hh"
#include "random/cuda/RngEngine.hh"
#include "physics/grid/XsCalculator.hh"
#include "DetectorView.hh"
#include "KernelUtils.hh"

using namespace celeritas;
using celeritas::detail::KleinNishinaInteractor;

namespace demo_interactor
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Kernel to initialize particle data.
 */
__global__ void initialize_kernel(ParamsDeviceRef const params,
                                  StateDeviceRef const  states,
                                  InitialPointers const init)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if out of range or already dead
    if (tid >= states.size())
    {
        return;
    }

    ParticleTrackView particle(params.particle, states.particle, ThreadId(tid));
    particle = init.particle;

    // Particles begin alive and in the +z direction
    states.direction[tid] = {0, 0, 1};
    states.position[tid]  = {0, 0, 0};
    states.time[tid]      = 0;
    states.alive[tid]     = true;
}

//---------------------------------------------------------------------------//
/*!
 * Sample cross sections and move to the collision point.
 */
__global__ void
move_kernel(ParamsDeviceRef const params, StateDeviceRef const states)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if out of range or already dead
    if (tid >= states.size() || !states.alive[tid])
    {
        return;
    }

    // Construct particle accessor from immutable and thread-local data
    ParticleTrackView particle(params.particle, states.particle, ThreadId(tid));
    RngEngine         rng(states.rng, ThreadId(tid));

    // Move to collision
    XsCalculator calc_xs(params.tables.xs, params.tables.reals);
    demo_interactor::move_to_collision(particle,
                                       calc_xs,
                                       states.direction[tid],
                                       &states.position[tid],
                                       &states.time[tid],
                                       rng);
}

//---------------------------------------------------------------------------//
/*!
 * Perform the iteraction plus cleanup.
 *
 * The interaction:
 * - Allocates and emits a secondary
 * - Kills the secondary, depositing its local energy
 * - Applies the interaction (updating track direction and energy)
 */
__global__ void interact_kernel(ParamsDeviceRef const            params,
                                StateDeviceRef const             states,
                                SecondaryAllocatorPointers const secondaries,
                                DetectorPointers const           detector)
{
    SecondaryAllocatorView allocate_secondaries(secondaries);
    unsigned int           tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if out of range or already dead
    if (tid >= states.size() || !states.alive[tid])
    {
        return;
    }

    // Construct particle accessor from immutable and thread-local data
    ParticleTrackView particle(params.particle, states.particle, ThreadId(tid));
    RngEngine         rng(states.rng, ThreadId(tid));

    DetectorView detector_hit(detector);
    Hit          h;
    h.pos    = states.position[tid];
    h.dir    = states.direction[tid];
    h.thread = ThreadId(tid);
    h.time   = states.time[tid];

    if (particle.energy() < units::MevEnergy{0.01})
    {
        // Particle is below interaction energy
        h.energy_deposited = particle.energy();

        // Deposit energy and kill
        detector_hit(h);
        states.alive[tid] = false;
        return;
    }

    // Construct RNG and interaction interfaces
    KleinNishinaInteractor interact(
        params.kn_interactor, particle, h.dir, allocate_secondaries);

    // Perform interaction: should emit a single particle (an electron)
    Interaction interaction = interact(rng);
    CELER_ASSERT(interaction);

    // Deposit energy from the secondary (effectively, an infinite energy
    // cutoff)
    {
        const auto& secondary = interaction.secondaries.front();
        h.dir                 = secondary.direction;
        h.energy_deposited    = secondary.energy;
        detector_hit(h);
    }

    // Update post-interaction state (apply interaction)
    states.direction[tid] = interaction.direction;
    particle.energy(interaction.energy);
}
} // namespace

//---------------------------------------------------------------------------//
// KERNEL INTERFACES
//---------------------------------------------------------------------------//
/*!
 * Initialize particle states.
 */
void initialize(const CudaGridParams&  opts,
                const ParamsDeviceRef& params,
                const StateDeviceRef&  states,
                const InitialPointers& initial)
{
    static const KernelParamCalculator calc_kernel_params(
        initialize_kernel, "initialize", opts.block_size);
    auto grid = calc_kernel_params(states.size());

    CELER_EXPECT(states.alive.size() == states.size());
    CELER_EXPECT(states.rng.size() == states.size());
    initialize_kernel<<<grid.grid_size, grid.block_size>>>(
        params, states, initial);
    CELER_CUDA_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
/*!
 * Run an iteration.
 */
void iterate(const CudaGridParams&              opts,
             const ParamsDeviceRef&             params,
             const StateDeviceRef&              states,
             const SecondaryAllocatorPointers&  secondaries,
             const celeritas::DetectorPointers& detector)
{
    static const KernelParamCalculator calc_kernel_params(
        move_kernel, "move", opts.block_size);
    auto grid = calc_kernel_params(states.size());

    move_kernel<<<grid.grid_size, grid.block_size>>>(params, states);
    CELER_CUDA_CHECK_ERROR();

    static const KernelParamCalculator calc_interact_params(
        interact_kernel, "interact", opts.block_size);
    grid = calc_interact_params(states.size());
    interact_kernel<<<grid.grid_size, grid.block_size>>>(
        params, states, secondaries, detector);
    CELER_CUDA_CHECK_ERROR();

    if (opts.sync)
    {
        // Note: the device synchronize is useful for debugging and necessary
        // for timing diagnostics.
        CELER_CUDA_CALL(hipDeviceSynchronize());
    }
}

//---------------------------------------------------------------------------//
/*!
 * Sum the total number of living particles.
 */
size_type reduce_alive(Span<bool> alive, const CudaGridParams& grid)
{
    size_type result = thrust::reduce(
        thrust::device_pointer_cast(alive.data()),
        thrust::device_pointer_cast(alive.data() + alive.size()),
        size_type(0),
        thrust::plus<size_type>());
    CELER_CUDA_CHECK_ERROR();

    if (grid.sync)
    {
        CELER_CUDA_CALL(hipDeviceSynchronize());
    }
    return result;
}

//---------------------------------------------------------------------------//
} // namespace demo_interactor
