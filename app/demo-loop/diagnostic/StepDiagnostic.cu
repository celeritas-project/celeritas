#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2021-2022 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file demo-loop/diagnostic/StepDiagnostic.cu
//---------------------------------------------------------------------------//
#include "StepDiagnostic.hh"

#include "corecel/sys/KernelParamCalculator.device.hh"

using namespace celeritas;

namespace demo_loop
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Count the steps per track for each particle type.
 */
__global__ void count_steps_kernel(CoreParamsDeviceRef const params,
                                   CoreStateDeviceRef const states,
                                   StepDiagnosticDataRef<MemSpace::device> data)
{
    auto tid = KernelParamCalculator::thread_id();
    if (!(tid < states.size()))
        return;

    StepLauncher<MemSpace::device> launch(params, states, data);
    launch(tid);
}

//---------------------------------------------------------------------------//
// KERNEL INTERFACES
//---------------------------------------------------------------------------//
/*!
 * Launch kernel to tally the steps per track.
 */
void count_steps(CoreParamsDeviceRef const& params,
                 CoreStateDeviceRef const& states,
                 StepDiagnosticDataRef<MemSpace::device> data)
{
    CELER_LAUNCH_KERNEL(count_steps,
                        celeritas::device().default_block_size(),
                        states.size(),
                        params,
                        states,
                        data);
}
//---------------------------------------------------------------------------//
}  // namespace demo_loop
