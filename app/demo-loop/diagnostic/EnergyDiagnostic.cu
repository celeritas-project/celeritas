#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
// Copyright 2021 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file EnergyDiagnostic.cu
//---------------------------------------------------------------------------//
#include "EnergyDiagnostic.hh"

#include "base/CollectionBuilder.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "base/Macros.hh"

namespace demo_loop
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Get energy deposition from state data and accumulate in appropriate bin
 */
__global__ void
bin_energy_kernel(const StateDataRefDevice states, PointersDevice pointers)
{
    auto tid = KernelParamCalculator::thread_id();
    if (!(tid < states.size()))
        return;

    EnergyDiagnosticLauncher<MemSpace::device> launch(states, pointers);
    launch(tid);
}

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
void bin_energy(const StateDataRefDevice& states, PointersDevice& pointers)
{
    static const celeritas::KernelParamCalculator calc_launch_params(
        bin_energy_kernel, "bin_energy");
    auto lparams = calc_launch_params(states.size());
    bin_energy_kernel<<<lparams.grid_size, lparams.block_size>>>(states,
                                                                 pointers);
    CELER_CUDA_CHECK_ERROR();
}

} // namespace demo_loop
