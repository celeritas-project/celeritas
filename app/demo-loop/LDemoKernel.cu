#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2021 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file LDemoKernel.cu
//---------------------------------------------------------------------------//
#include "LDemoKernel.hh"

#include "base/KernelParamCalculator.cuda.hh"
#include "random/RngEngine.hh"
#include "random/distributions/ExponentialDistribution.hh"
#include "physics/base/PhysicsStepUtils.hh"

using namespace celeritas;

namespace demo_loop
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
__global__ void
pre_step_kernel(ParamsDeviceRef const params, StateDeviceRef const states)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= states.size())
        return;

    CELER_NOT_IMPLEMENTED("Pre-step kernel");
#if 0
    PhysicsTrackView phys(inp.params, inp.states, init.particle, init.mat, tid);
    phys = PhysicsTrackInitializer{};

    // Sample mean free path
    {
        RngEngine                 rng(states.rng, ThreadId(tid));
        ExponentialDistribution<> sample_exponential;
        phys.interaction_mfp(sample_exponential(rng));
    }
#endif
}

} // namespace
//---------------------------------------------------------------------------//
// KERNEL INTERFACES
//---------------------------------------------------------------------------//
#define CDL_LAUNCH_KERNEL(NAME, THREADS, ARGS...)                   \
    do                                                              \
    {                                                               \
        static const ::celeritas::KernelParamCalculator NAME##_ckp( \
            NAME##_kernel, #NAME);                                  \
        auto kp = NAME##_ckp(THREADS);                              \
                                                                    \
        NAME##_kernel<<<kp.grid_size, kp.block_size>>>(ARGS);       \
        CELER_CUDA_CHECK_ERROR();                                   \
    } while (0)

//---------------------------------------------------------------------------//
void pre_step(const ParamsDeviceRef& params, const StateDeviceRef& states)
{
    CDL_LAUNCH_KERNEL(pre_step, states.size(), params, states);
}

//---------------------------------------------------------------------------//
} // namespace demo_loop
