#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2021-2022 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file orange/surfaces/SurfaceAction.test.cu
//---------------------------------------------------------------------------//
#include "SurfaceAction.test.hh"

#include "corecel/device_runtime_api.h"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "corecel/sys/Device.hh"

namespace celeritas_test
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void sa_test_kernel(SATestInput input)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= input.states.size())
        return;

    // Calculate distances in parallel
    CalcSenseDistanceLauncher<> calc_thread{input.params, input.states};
    calc_thread(tid);
}
} // namespace

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
void sa_test(SATestInput input)
{
    CELER_LAUNCH_KERNEL(sa_test,
                        celeritas::device().default_block_size(),
                        input.states.size(),
                        input);
    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
