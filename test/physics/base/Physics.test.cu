#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2021 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file Physics.test.cu
//---------------------------------------------------------------------------//
#include "Physics.test.hh"

#include "base/KernelParamCalculator.cuda.hh"

using namespace celeritas;

namespace celeritas_test
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void phys_test_kernel(const PTestInput inp)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= inp.states.size())
        return;

    const auto&      init = inp.inits[tid];
    PhysicsTrackView phys(inp.params, inp.states, init.particle, init.mat, tid);

    phys                  = PhysicsTrackInitializer{};
    inp.result[tid.get()] = calc_step(phys, init.energy);
}
} // namespace

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
void phys_cuda_test(const PTestInput& input)
{
    CELER_ASSERT(input.inits.size() == input.states.size());
    static const KernelParamCalculator calc_launch_params(phys_test_kernel,
                                                          "phys_test");
    auto params = calc_launch_params(input.states.size());
    phys_test_kernel<<<params.grid_size, params.block_size>>>(input);

    CELER_CUDA_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
