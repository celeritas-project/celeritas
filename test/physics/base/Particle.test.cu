#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file Particle.test.cu
//---------------------------------------------------------------------------//
#include "physics/base/ParticleTrackView.hh"
#include "Particle.test.hh"

#include <thrust/device_vector.h>
#include "base/KernelParamCalculator.cuda.hh"

using thrust::raw_pointer_cast;

namespace celeritas_test
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void ptv_test_kernel(unsigned int              size,
                                ParticleParamsPointers    params,
                                ParticleStatePointers     states,
                                const ParticleTrackState* init,
                                double*                   result)
{
    auto local_thread_id = celeritas::KernelParamCalculator::thread_id();
    if (!(local_thread_id < size))
        return;

    // Initialize particle
    ParticleTrackView p(params, states, local_thread_id);
    p = init[local_thread_id.get()];

    // Skip result to the start for this thread
    result += local_thread_id.get() * PTVTestOutput::props_per_thread();

    // Calculate/write values from the track view
    CELER_ASSERT(p.particle_id() == init[local_thread_id.get()].particle_id);
    *result++ = p.energy().value();
    *result++ = p.mass().value();
    *result++ = p.charge().value();
    *result++ = p.decay_constant();
    *result++ = p.speed().value();
    *result++ = (p.mass() > zero_quantity() ? p.lorentz_factor() : -1);
    *result++ = p.momentum().value();
    *result++ = p.momentum_sq().value();
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
PTVTestOutput ptv_test(PTVTestInput input)
{
    thrust::device_vector<ParticleTrackState> init = input.init;
    thrust::device_vector<double>             result(init.size()
                                         * PTVTestOutput::props_per_thread());

    static const celeritas::KernelParamCalculator calc_launch_params(
        ptv_test_kernel, "ptv_test");
    auto params = calc_launch_params(init.size());
    ptv_test_kernel<<<params.grid_size, params.block_size>>>(
        init.size(),
        input.params,
        input.states,
        raw_pointer_cast(init.data()),
        raw_pointer_cast(result.data()));
    CELER_CUDA_CHECK_ERROR();
    CELER_CUDA_CALL(hipDeviceSynchronize());

    PTVTestOutput output;
    output.props.resize(result.size());
    thrust::copy(result.begin(), result.end(), output.props.begin());
    return output;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
