#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file TrackInit.test.cu
//---------------------------------------------------------------------------//
#include "TrackInit.test.hh"

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include "base/KernelParamCalculator.cuda.hh"

namespace celeritas_test
{
using namespace celeritas;

//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void interact_kernel(StateDeviceRef             states,
                                SecondaryAllocatorPointers secondaries,
                                ITTestInputPointers        input)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id < states.size())
    {
        SimTrackView sim(states.sim, thread_id);

        // There may be more track slots than active tracks; only active tracks
        // should interact
        if (sim.alive())
        {
            // Allow the particle to interact and create secondaries
            StackAllocator<Secondary> allocate_secondaries(secondaries);
            Interactor                interact(allocate_secondaries,
                                input.alloc_size[thread_id.get()],
                                input.alive[thread_id.get()]);
            states.interactions[thread_id] = interact();

            // Kill the selected tracks
            if (!input.alive[thread_id.get()])
            {
                sim.alive(false);
            }
        }
        else
        {
            states.interactions[thread_id] = Interaction::from_absorption();
        }
    }
}

__global__ void tracks_test_kernel(StateDeviceRef states, unsigned int* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id < states.size())
    {
        SimTrackView sim(states.sim, thread_id);
        output[thread_id.get()] = sim.track_id().get();
    }
}

__global__ void
initializers_test_kernel(TrackInitStateDeviceRef inits, unsigned int* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id < inits.initializers.size())
    {
        TrackInitializer& init  = inits.initializers[thread_id];
        output[thread_id.get()] = init.sim.track_id.get();
    }
}

__global__ void
vacancies_test_kernel(TrackInitStateDeviceRef inits, size_type* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id < inits.vacancies.size())
    {
        output[thread_id.get()] = inits.vacancies[thread_id];
    }
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//

void interact(StateDeviceRef             states,
              SecondaryAllocatorPointers secondaries,
              ITTestInputPointers        input)
{
    CELER_EXPECT(states.size() > 0);
    CELER_EXPECT(states.size() == input.alloc_size.size());

    static const KernelParamCalculator calc_launch_params(interact_kernel,
                                                          "interact");
    auto lparams = calc_launch_params(states.size());
    interact_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, secondaries, input);
    CELER_CUDA_CHECK_ERROR();
}

std::vector<unsigned int> tracks_test(StateDeviceRef states)
{
    // Allocate memory for results
    std::vector<unsigned int> host_output(states.size());
    if (states.size() == 0)
    {
        return host_output;
    }
    thrust::device_vector<unsigned int> output(states.size());

    // Launch a kernel to check the track ID of the initialized tracks
    static const celeritas::KernelParamCalculator calc_launch_params(
        tracks_test_kernel, "tracks_test");
    auto lparams = calc_launch_params(states.size());
    tracks_test_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, thrust::raw_pointer_cast(output.data()));
    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

std::vector<unsigned int> initializers_test(TrackInitStateDeviceRef inits)
{
    // Allocate memory for results
    std::vector<unsigned int> host_output(inits.initializers.size());
    if (inits.initializers.size() == 0)
    {
        return host_output;
    }
    thrust::device_vector<unsigned int> output(inits.initializers.size());

    // Launch a kernel to check the track ID of the track initializers
    static const celeritas::KernelParamCalculator calc_launch_params(
        initializers_test_kernel, "initializers_test");
    auto lparams = calc_launch_params(inits.initializers.size());
    initializers_test_kernel<<<lparams.grid_size, lparams.block_size>>>(
        inits, thrust::raw_pointer_cast(output.data()));
    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

std::vector<size_type> vacancies_test(TrackInitStateDeviceRef inits)
{
    // Allocate memory for results
    std::vector<size_type> host_output(inits.vacancies.size());
    if (inits.vacancies.size() == 0)
    {
        return host_output;
    }
    thrust::device_vector<size_type> output(inits.vacancies.size());

    // Launch a kernel to check the indices of the empty slots
    static const celeritas::KernelParamCalculator calc_launch_params(
        vacancies_test_kernel, "vacancies_test");
    auto lparams = calc_launch_params(inits.vacancies.size());
    vacancies_test_kernel<<<lparams.grid_size, lparams.block_size>>>(
        inits, thrust::raw_pointer_cast(output.data()));
    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
