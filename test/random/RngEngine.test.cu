#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2021-2022 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file RngEngine.test.cu
//---------------------------------------------------------------------------//

#include "random/RngEngine.hh"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include "base/device_runtime_api.h"
#include "base/KernelParamCalculator.device.hh"
#include "comm/Device.hh"

#include "RngEngine.test.hh"

using namespace celeritas;
using thrust::raw_pointer_cast;

namespace celeritas_test
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void
sample_native_kernel(RngStateData<Ownership::reference, MemSpace::device> view,
                     RngEngine::result_type* samples)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() < view.size())
    {
        RngEngine rng(view, tid);
        samples[tid.get()] = rng();
    }
}

template<class RealType>
__global__ void
sample_canonical_kernel(RngStateData<Ownership::reference, MemSpace::device> view,
                        RealType* samples)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() < view.size())
    {
        RngEngine rng(view, tid);
        samples[tid.get()] = generate_canonical<RealType>(rng);
    }
}
} // namespace

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
std::vector<unsigned int> re_test_native(RngDeviceRef states)
{
    thrust::device_vector<unsigned int> samples(states.size());

    CELER_LAUNCH_KERNEL(sample_native,
                        celeritas::device().default_block_size(),
                        states.size(),
                        states,
                        raw_pointer_cast(samples.data()));

    std::vector<unsigned int> host_samples(states.size());
    thrust::copy(samples.begin(), samples.end(), host_samples.begin());

    return host_samples;
}

//---------------------------------------------------------------------------//
//! Run on device and return results
template<class T>
std::vector<T> re_test_canonical(RngDeviceRef states)
{
    thrust::device_vector<T> samples(states.size());

    static const ::celeritas::KernelParamCalculator calc_launch_params(
        sample_canonical_kernel<T>,
        "sample_canonical",
        celeritas::device().default_block_size());
    auto grid = calc_launch_params(states.size());

    CELER_LAUNCH_KERNEL_IMPL(sample_canonical_kernel<T>,
                             grid.grid_size,
                             grid.block_size,
                             0,
                             0,
                             states,
                             raw_pointer_cast(samples.data()));
    CELER_DEVICE_CHECK_ERROR();
    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());

    std::vector<T> host_samples(states.size());
    thrust::copy(samples.begin(), samples.end(), host_samples.begin());

    return host_samples;
}

//---------------------------------------------------------------------------//
// EXPLICIT INSTANTIATION
//---------------------------------------------------------------------------//

template std::vector<float>  re_test_canonical<float>(RngDeviceRef);
template std::vector<double> re_test_canonical<double>(RngDeviceRef);

//---------------------------------------------------------------------------//
} // namespace celeritas_test
