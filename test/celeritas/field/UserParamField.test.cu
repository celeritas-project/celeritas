#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020-2022 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/field/UserParamField.test.cu
//---------------------------------------------------------------------------//
#include <thrust/device_vector.h>

#include "corecel/device_runtime_api.h"
#include "corecel/Types.hh"
#include "corecel/cont/Range.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/Constants.hh"
#include "celeritas/field/DormandPrinceStepper.hh"
#include "celeritas/field/FieldDriver.hh"
#include "celeritas/field/FieldParamsData.hh"
#include "celeritas/field/FieldPropagator.hh"
#include "celeritas/field/MagFieldEquation.hh"
#include "celeritas/field/MagFieldTraits.hh"
#include "celeritas/geo/GeoTrackView.hh"
#include "celeritas/phys/ParticleTrackView.hh"

#include "FieldPropagator.test.hh"
#include "FieldTestParams.hh"
#include "UserField.test.hh"
#include "detail/CMSParameterizedField.hh"

using celeritas_test::detail::CMSParameterizedField;
using thrust::raw_pointer_cast;

namespace celeritas_test
{
using namespace celeritas;
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void parameterized_field_test_kernel(UserFieldTestParams param,
                                                real_type*          value_x,
                                                real_type*          value_y,
                                                real_type*          value_z)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= param.nsamples)
        return;

    detail::CMSParameterizedField field;
    Real3                         pos{tid.get() * param.delta_r,
              tid.get() * param.delta_r,
              tid.get() * param.delta_z};
    Real3                         value = field(pos);

    // Output for verification
    value_x[tid.get()] = value[0];
    value_y[tid.get()] = value[1];
    value_z[tid.get()] = value[2];
}

__global__ void par_fp_test_kernel(const int                  size,
                                   const GeoParamsCRefDevice  shared,
                                   const GeoStateRefDevice    state,
                                   const GeoTrackInitializer* start,
                                   const ParticleParamsRef    particle_params,
                                   ParticleStateRef           particle_states,
                                   FieldParamsData            field_params,
                                   FieldTestParams            test,
                                   const ParticleTrackInitializer* init_track,
                                   double*                         pos,
                                   double*                         dir,
                                   double*                         step)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= size)
        return;

    // Initialize GeoTrackView and ParticleTrackView
    GeoTrackView geo_track(shared, state, tid);
    geo_track = start[tid.get()];
    if (!geo_track.is_outside())
        geo_track.find_next_step();

    ParticleTrackView particle_track(particle_params, particle_states, tid);
    particle_track = init_track[tid.get()];

    // Construct the field propagator with a user CMSParameterizedField
    CMSParameterizedField field;
    using MFTraits
        = MagFieldTraits<CMSParameterizedField, DormandPrinceStepper>;
    MFTraits::Equation_t   equation(field, units::ElementaryCharge{-1});
    MFTraits::Stepper_t    stepper(equation);
    MFTraits::Driver_t     driver(field_params, &stepper);
    MFTraits::Propagator_t propagator(particle_track, &geo_track, &driver);

    // Tests with input parameters of a electron in a uniform magnetic field
    double hstep = (2.0 * constants::pi * test.radius) / test.nsteps;

    real_type curved_length = 0;

    MFTraits::Propagator_t::result_type result;

    for (CELER_MAYBE_UNUSED int i : celeritas::range(test.revolutions))
    {
        for (CELER_MAYBE_UNUSED int j : celeritas::range(test.nsteps))
        {
            result = propagator(hstep);
            curved_length += result.distance;
            CELER_ASSERT(!result.boundary);
        }
    }

    // output
    step[tid.get()] = curved_length;
}

__global__ void par_bc_test_kernel(const int                  size,
                                   const GeoParamsCRefDevice  shared,
                                   const GeoStateRefDevice    state,
                                   const GeoTrackInitializer* start,
                                   ParticleParamsRef          particle_params,
                                   ParticleStateRef           particle_states,
                                   FieldParamsData            field_params,
                                   FieldTestParams            test,
                                   const ParticleTrackInitializer* init_track,
                                   double*                         pos,
                                   double*                         dir,
                                   double*                         step)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= size)
        return;

    // Initialize GeoTrackView and ParticleTrackView
    GeoTrackView geo_track(shared, state, tid);
    geo_track = start[tid.get()];
    if (!geo_track.is_outside())
        geo_track.find_next_step();

    ParticleTrackView particle_track(particle_params, particle_states, tid);
    particle_track = init_track[tid.get()];

    // Construct the field propagator with a user CMSParameterizedField
    CMSParameterizedField field;
    using MFTraits
        = MagFieldTraits<CMSParameterizedField, DormandPrinceStepper>;
    MFTraits::Equation_t   equation(field, units::ElementaryCharge{-1});
    MFTraits::Stepper_t    stepper(equation);
    MFTraits::Driver_t     driver(field_params, &stepper);
    MFTraits::Propagator_t propagator(particle_track, &geo_track, &driver);

    // Tests with input parameters of a electron in a uniform magnetic field
    double hstep = (2.0 * constants::pi * test.radius) / test.nsteps;

    real_type curved_length = 0;

    constexpr int num_boundary = 4;
    int           icross       = 0;

    constexpr real_type expected_y[num_boundary] = {0.5, 0.5, -0.5, -0.5};

    real_type delta = celeritas::numeric_limits<real_type>::max();

    MFTraits::Propagator_t::result_type result;

    for (CELER_MAYBE_UNUSED int ir : celeritas::range(test.revolutions))
    {
        for (CELER_MAYBE_UNUSED int i : celeritas::range(test.nsteps))
        {
            result = propagator(hstep);
            curved_length += result.distance;

            if (result.boundary)
            {
                icross++;
                int j = (icross - 1) % num_boundary;
                delta = expected_y[j] - geo_track.pos()[1];
                if (delta != 0)
                {
                    printf("Intersection Finding Failed on GPU: ");
                    printf("Expected = %f Actual = %f\n",
                           expected_y[j],
                           geo_track.pos()[1]);
                }
                geo_track.cross_boundary();
            }
        }
    }

    // output
    step[tid.get()] = curved_length;
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
UserFieldTestOutput parameterized_field_test(UserFieldTestParams test_param)
{
    // Output data for kernel
    thrust::device_vector<real_type> value_x(test_param.nsamples, 0.0);
    thrust::device_vector<real_type> value_y(test_param.nsamples, 0.0);
    thrust::device_vector<real_type> value_z(test_param.nsamples, 0.0);

    // Run kernel
    CELER_LAUNCH_KERNEL(parameterized_field_test,
                        celeritas::device().default_block_size(),
                        test_param.nsamples,
                        test_param,
                        raw_pointer_cast(value_x.data()),
                        raw_pointer_cast(value_y.data()),
                        raw_pointer_cast(value_z.data()));
    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());

    // Copy result back to CPU
    UserFieldTestOutput result;

    result.value_x.resize(value_x.size());
    thrust::copy(value_x.begin(), value_x.end(), result.value_x.begin());

    result.value_y.resize(value_y.size());
    thrust::copy(value_y.begin(), value_y.end(), result.value_y.begin());

    result.value_z.resize(value_z.size());
    thrust::copy(value_z.begin(), value_z.end(), result.value_z.begin());

    return result;
}

//! Run on device and return results
UserFieldTestVector par_fp_test(FPTestInput input)
{
    CELER_ASSERT(input.init_geo.size() == input.init_track.size());
    CELER_ASSERT(input.geo_params);
    CELER_ASSERT(input.geo_states);

    // Temporary device data for kernel
    thrust::device_vector<GeoTrackInitializer> in_geo(input.init_geo.begin(),
                                                      input.init_geo.end());
    thrust::device_vector<ParticleTrackInitializer> in_track = input.init_track;

    // Output data for kernel
    thrust::device_vector<double> step(input.init_geo.size(), -1.0);
    thrust::device_vector<double> pos(input.init_geo.size(), -1.0);
    thrust::device_vector<double> dir(input.init_geo.size(), -1.0);

    // Run kernel
    CELER_LAUNCH_KERNEL(par_fp_test,
                        celeritas::device().default_block_size(),
                        in_geo.size(),
                        in_geo.size(),
                        input.geo_params,
                        input.geo_states,
                        raw_pointer_cast(in_geo.data()),
                        input.particle_params,
                        input.particle_states,
                        input.field_params,
                        input.test,
                        raw_pointer_cast(in_track.data()),
                        raw_pointer_cast(pos.data()),
                        raw_pointer_cast(dir.data()),
                        raw_pointer_cast(step.data()));
    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());

    // Copy result back to CPU
    UserFieldTestVector result;

    result.resize(step.size());
    thrust::copy(step.begin(), step.end(), result.begin());

    return result;
}

//! Run a boundary crossing test on device and return results
UserFieldTestVector par_bc_test(FPTestInput input)
{
    CELER_ASSERT(input.init_geo.size() == input.init_track.size());
    CELER_ASSERT(input.geo_params);
    CELER_ASSERT(input.geo_states);

    // Temporary device data for kernel
    thrust::device_vector<GeoTrackInitializer> in_geo(input.init_geo.begin(),
                                                      input.init_geo.end());
    thrust::device_vector<ParticleTrackInitializer> in_track = input.init_track;

    // Output data for kernel
    thrust::device_vector<double> step(input.init_geo.size(), -1.0);
    thrust::device_vector<double> pos(input.init_geo.size(), -1.0);
    thrust::device_vector<double> dir(input.init_geo.size(), -1.0);

    // Run kernel
    CELER_LAUNCH_KERNEL(par_bc_test,
                        celeritas::device().default_block_size(),
                        in_geo.size(),
                        in_geo.size(),
                        input.geo_params,
                        input.geo_states,
                        raw_pointer_cast(in_geo.data()),
                        input.particle_params,
                        input.particle_states,
                        input.field_params,
                        input.test,
                        raw_pointer_cast(in_track.data()),
                        raw_pointer_cast(pos.data()),
                        raw_pointer_cast(dir.data()),
                        raw_pointer_cast(step.data()));
    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());

    // Copy result back to CPU
    UserFieldTestVector result;

    result.resize(step.size());
    thrust::copy(step.begin(), step.end(), result.begin());

    return result;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
