#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
// Copyright 2020-2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/field/DormandPrinceStepper.cu
//---------------------------------------------------------------------------//
#include "DormandPrinceStepper.test.hh"

#include <typeinfo>

#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "corecel/sys/ScopedProfiling.hh"
#include "celeritas/field/detail/FieldUtils.hh"

namespace celeritas
{
namespace test
{
//---------------------------------------------------------------------------//

namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
template<class Stepper_impl>
__device__ FieldStepperResult run_stepper(Stepper_impl& stepper,
                                          int step,
                                          OdeState state,
                                          int number_threads,
                                          int number_states,
                                          OdeState* ks,
                                          OdeState* along_states,
                                          FieldStepperResult* result);

template<>
__device__ FieldStepperResult run_stepper(StepperUni& stepper,
                                          int step,
                                          OdeState state,
                                          int number_threads,
                                          int number_states,
                                          OdeState* ks,
                                          OdeState* along_states,
                                          FieldStepperResult* result)
{
    return stepper(step, state);
}

template<>
__device__ FieldStepperResult run_stepper(StepperMultiGlobal& stepper,
                                          int step,
                                          OdeState state,
                                          int number_threads,
                                          int number_states,
                                          OdeState* ks,
                                          OdeState* along_states,
                                          FieldStepperResult* result)
{
    return stepper(step, state, number_threads, ks, along_states, result);
}

template<>
__device__ FieldStepperResult run_stepper(StepperMultiShared& stepper,
                                          int step,
                                          OdeState state,
                                          int number_threads,
                                          int number_states,
                                          OdeState* ks,
                                          OdeState* along_states,
                                          FieldStepperResult* result)
{
    return stepper(step, state, number_threads, number_states);
}

template<class Stepper_impl>
__global__ void dormand_test_arg_kernel(OdeState* states,
                                        FieldStepperResult* results,
                                        int number_iterations,
                                        int number_threads,
                                        int number_states,
                                        OdeState* ks,
                                        OdeState* along_states)
{
    constexpr double initial_step_size = 10000.0;
    constexpr double delta_chord = 1e-4;
    constexpr double half = 0.5;

    auto id = KernelParamCalculator::thread_id().get() / number_threads;
    if (id >= number_states)
    {
        return;
    }

    auto eval = make_dummy_equation(dormand_prince_dummy_field);
    Stepper_impl stepper{eval};
    FieldStepperResult res;
    auto state = states[id];
    auto step = initial_step_size;

    for (int i = 0; i < number_iterations; ++i)
    {
        res = run_stepper(stepper,
                          step,
                          state,
                          number_threads,
                          number_states,
                          &ks[id * 7],
                          &along_states[id],
                          &results[id]);
        auto dchord
            = detail::distance_chord(state, res.mid_state, res.end_state);
        step *= max(std::sqrt(delta_chord / dchord), half);
    }
    results[id] = res;
}
}  // namespace

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
KernelResult simulate_multi_next_chord(int number_threads,
                                       int number_states,
                                       bool use_shared)
{
    KernelResult result;
    bool is_global_version = number_threads > 1 && !use_shared;

    // Load initial states and results to device
    FieldStepperResult* d_results;
    std::vector<FieldStepperResult> h_results(number_states);

    OdeState* d_along_states;
    std::vector<OdeState> h_along_states(number_states);

    //TODO: Move this into "build_variables"
    OdeState *h_ks, *d_ks;
    if (is_global_version)
    {
        h_ks = new OdeState[number_states * 7];
        for (int i = 0; i < number_states * 7; ++i)
        {
            h_ks[i] = OdeState();
        }
    }

    OdeState *h_states, *d_states;
    h_states = new OdeState[number_states];

    build_variables(number_states, is_global_version,
    h_results.data(), h_along_states.data(), h_states);

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate memory on device
    hipMalloc(&d_results, number_states * sizeof(FieldStepperResult));
    hipMalloc(&d_states, number_states * sizeof(OdeState));
    if (is_global_version)
    {
        hipMalloc(&d_ks, number_states * 7 * sizeof(OdeState));
        hipMalloc(&d_along_states, number_states * sizeof(OdeState));
    }

    // Copy initial states to device
    hipMemcpy(d_states,
               h_states,
               number_states * sizeof(OdeState),
               hipMemcpyHostToDevice);
    if (is_global_version)
    {
        hipMemcpy(d_ks,
                   h_ks,
                   number_states * 7 * sizeof(OdeState),
                   hipMemcpyHostToDevice);
        hipMemcpy(d_along_states,
                   h_along_states.data(),
                   number_states * sizeof(OdeState),
                   hipMemcpyHostToDevice);
    }

    // Kernel configuration
    constexpr int max_threads_per_block = 768;
    int grid_size = number_threads * number_states / max_threads_per_block;
    grid_size += (number_threads * number_states) % max_threads_per_block
                    == 0
        ? 0
        : 1;
    int nb_warps = number_threads * number_states / (32 * grid_size);
    nb_warps += (number_threads * number_states) % (32 * grid_size) == 0 ? 0 : 1;
    int block_size = nb_warps * 32;
    int shared_memory = 0;
    if (use_shared)
    {
        constexpr int max_threads_per_block_shared = 320;
        grid_size = number_threads * number_states / max_threads_per_block_shared;
        grid_size += (number_threads * number_states) % max_threads_per_block_shared
                        == 0
            ? 0
            : 1;
        nb_warps = number_threads * number_states / (32 * grid_size);
        nb_warps += (number_threads * number_states) % (32 * grid_size) == 0 ? 0 : 1;
        block_size = nb_warps * 32;

        shared_memory = (block_size / 4) * 7 * sizeof(OdeState)
                        + (block_size / 4) * sizeof(OdeState)
                        + (block_size / 4) * sizeof(FieldStepperResult);
    }

    // Launch the kernel with the desired streamId
    hipEventRecord(start);
    if (number_threads > 1)
    {
        if (use_shared)
        {
            dormand_test_arg_kernel<StepperMultiShared>
                <<<grid_size, block_size, shared_memory>>>(
                    d_states,
                    d_results,
                    number_iterations,
                    number_threads,
                    number_states,
                    d_ks,
                    d_along_states);
        }
        else
        {
            dormand_test_arg_kernel<StepperMultiGlobal>
                <<<grid_size, block_size>>>(d_states,
                                                        d_results,
                                                        number_iterations,
                                                        number_threads,
                                                        number_states,
                                                        d_ks,
                                                        d_along_states);
        }
    }
    else
    {
        dormand_test_arg_kernel<StepperUni>
            <<<grid_size, block_size>>>(d_states,
                                                    d_results,
                                                    number_iterations,
                                                    number_threads,
                                                        number_states,
                                                    d_ks,
                                                    d_along_states);
    }

    hipDeviceSynchronize();
    hipEventRecord(stop);

    // Check if kernel execution generated an error
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        int max_threads_per_block = 0;
        int max_blocks = 0;
        int max_shared_memory = 0;

        hipDeviceGetAttribute(
            &max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, 0);
        hipDeviceGetAttribute(&max_blocks, hipDeviceAttributeMaxGridDimX, 0);
        hipDeviceGetAttribute(
            &max_shared_memory, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        std::cerr << "Launch config for " << number_states
                  << " states: " << grid_size << " blocks, "
                  << block_size << " threads per block, "
                  << shared_memory << " bytes of "
                  << "shared memory"<< std::endl;
        std::cerr << "Device properties: " << max_threads_per_block
                  << " threads per block, " << max_blocks << " blocks and "
                  << max_shared_memory << " bytes of shared memory"
                  << std::endl;
        result.milliseconds = -1;
    }
    else
    {
        // Compute the elapsed time
        hipDeviceSynchronize();
        hipEventElapsedTime(&(result.milliseconds), start, stop);
    }

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy results back to hostComp
    hipMemcpy(h_results.data(),
               d_results,
               number_states * sizeof(FieldStepperResult),
               hipMemcpyDeviceToHost);

    // Free memory on device
    hipFree(d_results);
    hipFree(d_states);
    if (is_global_version)
    {
        hipFree(d_ks);
        hipFree(d_along_states);
    }

    // Return results
    result.results = h_results.data();
    return result;
}
//---------------------------------------------------------------------------//
}  // namespace test
}  // namespace celeritas
