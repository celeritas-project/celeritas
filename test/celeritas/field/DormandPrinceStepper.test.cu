#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
// Copyright 2020-2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/field/DormandPrinceStepper.cu
//---------------------------------------------------------------------------//
#include "DormandPrinceStepper.test.hh"

#include "corecel/sys/KernelParamCalculator.device.hh" // for CELER_LAUNCH_KERNEL
#include "corecel/sys/Device.hh" // device()
#include "corecel/sys/ScopedProfiling.hh" // ScopedProfiling
#include "celeritas/field/detail/FieldUtils.hh"

#include <typeinfo>

namespace celeritas
{
namespace test
{
//---------------------------------------------------------------------------//

namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
__global__ void test_kernel()
{
    // printf("Hello from block %d and thread %d\n", blockIdx.x, threadIdx.x);
    int i = threadIdx.x;
    int j = i;
    if (i < 4) {
        j = __shfl_down_sync(0x0000000f, i, 2);
        __syncthreads();
    }
    printf("Thread %d: before %d, after %d\n", threadIdx.x, i, j);
}

template<class Stepper_impl>
__device__ FieldStepperResult run_stepper(Stepper_impl stepper, int step, OdeState state, int id, int index, OdeState *ks, OdeState *along_state, FieldStepperResult *result)
{
    return FieldStepperResult();
}
  
template<>
__device__ FieldStepperResult run_stepper(Stepper_uni stepper, int step, OdeState state, int id, int index, OdeState *ks, OdeState *along_state, FieldStepperResult *result)
{
    if (index != 0) return FieldStepperResult();
    return stepper(step, state);
}

template<>
__device__ FieldStepperResult run_stepper(Stepper_multi stepper, int step, OdeState state, int id, int index, OdeState *ks, OdeState *along_state, FieldStepperResult *result)
{
    // printf("thread %d, index %d\n", id, index);

    return stepper(step, state, id, index, ks, along_state, result);
}

template<class Stepper_impl>
__global__ void dormand_test_arg_kernel(OdeState *states,
                                        FieldStepperResult *results,
                                        int *num_states, int *number_iterations,
                                        int *number_threads, OdeState *ks, OdeState *along_state)
{
    constexpr double initial_step_size = 10000.0;
    constexpr double delta_chord = 1e-4;
    constexpr double half = 0.5;

    auto id = (blockIdx.x * blockDim.x + threadIdx.x) / *number_threads;

    if (id >= *num_states) return;

    auto index = (blockIdx.x * blockDim.x + threadIdx.x) % *number_threads;
    auto eval = make_dummy_equation(dormand_prince_dummy_field);
    Stepper_impl stepper{eval};
    FieldStepperResult res;
    auto state = states[id];
    auto step = initial_step_size;

    for (int i = 0; i < *number_iterations; ++i)
    {
        res = run_stepper(stepper, step, state, id, index, &ks[id*7], &along_state[id], &results[id]);
        auto dchord = detail::distance_chord(state, res.mid_state, res.end_state);
        step *= max(std::sqrt(delta_chord / dchord), half);
    }
    results[id] = res;
}
} // namespace

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
void test()
{
    test_kernel<<<1, 32>>>();
}

KernelResult simulate_multi_next_chord(int number_threads)
{
    KernelResult result;

    // Load initial states and results to device
    int *d_num_states, *d_number_iterations, *d_number_threads;

    FieldStepperResult *h_results, *d_results;
    h_results = new FieldStepperResult[number_of_states];
    for (int i = 0; i < number_of_states; ++i)
    {
        h_results[i] = FieldStepperResult();
    }

    OdeState *h_along_state, *d_along_state, *d_states;
    h_along_state = new OdeState[number_of_states];
    for (int i = 0; i < number_of_states; ++i)
    {
        h_along_state[i] = OdeState();
    }

    OdeState *h_ks, *d_ks;
    h_ks = new OdeState[number_of_states * 7];
    for (int i = 0; i < number_of_states * 7; ++i)
    {
        h_ks[i] = OdeState();
    }

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate memory on device
    hipMalloc(&d_results, number_of_states * sizeof(FieldStepperResult));
    hipMalloc(&d_states, number_of_states * sizeof(OdeState));
    hipMalloc(&d_num_states, sizeof(int));
    hipMalloc(&d_number_iterations, sizeof(int));
    hipMalloc(&d_number_threads, sizeof(int));
    hipMalloc(&d_ks, number_of_states * 7 * sizeof(OdeState));
    hipMalloc(&d_along_state, number_of_states * sizeof(OdeState));

    // Copy initial states to device
    hipMemcpy(d_states, initial_states, number_of_states * sizeof(OdeState), hipMemcpyHostToDevice);
    hipMemcpy(d_num_states, &number_of_states, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_number_iterations, &number_iterations, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_number_threads, &number_threads, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ks, h_ks, number_of_states * 7 * sizeof(OdeState), hipMemcpyHostToDevice);
    hipMemcpy(d_along_state, &h_along_state, number_of_states * sizeof(OdeState), hipMemcpyHostToDevice);

    // Launch the kernel with the desired streamId
    // ScopedProfiling profile_this{"Dormand-Prince-test"};
    hipEventRecord(start);
    // CELER_LAUNCH_KERNEL(dormand_test_arg,
    //                     device().threads_per_warp(), number_of_states, 0,
    //                     d_states, d_results, d_num_states);
    if (number_threads > 1){
        dormand_test_arg_kernel<Stepper_multi><<<1, number_of_states * number_threads>>>
            (d_states, d_results, d_num_states, d_number_iterations, d_number_threads, d_ks, d_along_state);
    } else {
        dormand_test_arg_kernel<Stepper_uni><<<1, number_of_states>>>
            (d_states, d_results, d_num_states, d_number_iterations, d_number_threads, d_ks, d_along_state);
    }

    hipDeviceSynchronize();
    hipEventRecord(stop);

    // Compute the elapsed time
    hipDeviceSynchronize();
    hipEventElapsedTime(&(result.milliseconds), start, stop);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Copy results back to host
    hipMemcpy(h_results, d_results, number_of_states * sizeof(FieldStepperResult), hipMemcpyDeviceToHost);

    // Free memory on device
    hipFree(d_results);
    hipFree(d_states);
    hipFree(d_num_states);
    hipFree(d_number_iterations);
    hipFree(d_number_threads);
    hipFree(d_ks);
    hipFree(d_along_state);

    // Return results
    result.results = h_results;
    return result;
}
//---------------------------------------------------------------------------//
}  // namespace test
}  // namespace celeritas
