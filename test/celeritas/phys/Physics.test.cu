#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2021-2024 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/phys/Physics.test.cu
//---------------------------------------------------------------------------//
#include "Physics.test.hh"

#include "corecel/device_runtime_api.hh"

#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/phys/PhysicsStepView.hh"
#include "celeritas/phys/PhysicsTrackView.hh"
#include "celeritas/Quantities.hh"

namespace celeritas
{
namespace test
{
//---------------------------------------------------------------------------//

namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void phys_test_kernel(PTestInput const inp)
{
    auto tid = TrackSlotId{KernelParamCalculator::thread_id().unchecked_get()};
    if (tid.get() >= inp.states.size())
        return;

    auto const& init = inp.inits[tid];
    PhysicsTrackView phys(inp.params, inp.states, init.particle, init.mat, tid);
    PhysicsStepView step(inp.params, inp.states, tid);

    phys = PhysicsTrackInitializer{};
    inp.result[tid.get()]
        = native_value_to<units::CmLength>(calc_step(phys, step, init.energy))
              .value();
}
}  // namespace

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
void phys_cuda_test(PTestInput const& input)
{
    CELER_ASSERT(input.inits.size() == input.states.size());

    CELER_LAUNCH_KERNEL(phys_test, input.states.size(), 0, input);
}

//---------------------------------------------------------------------------//
}  // namespace test
}  // namespace celeritas
