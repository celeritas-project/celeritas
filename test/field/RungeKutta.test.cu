#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file RungeKutta.test.cu
//---------------------------------------------------------------------------//
#include "RungeKutta.test.hh"

#include "base/KernelParamCalculator.cuda.hh"
#include <thrust/device_vector.h>

#include "field/MagField.hh"
#include "field/FieldEquation.hh"
#include "field/RungeKuttaStepper.hh"

#include "base/Range.hh"
#include "base/Types.hh"
#include "base/Constants.hh"

using thrust::raw_pointer_cast;

namespace celeritas_test
{
using namespace celeritas;
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void rk4_test_kernel(FieldTestParams param,
                                real_type*      pos_x,
                                real_type*      pos_z,
                                real_type*      mom_y,
                                real_type*      mom_z,
                                real_type*      error)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= param.nstates)
        return;

    // Construct the Runge-Kutta stepper
    MagField                         field({0, 0, param.field_value});
    FieldEquation                    equation(field);
    RungeKuttaStepper<FieldEquation> rk4(equation);

    // Initial state and the epected state after revolutions
    //    OdeArray<real_type, 6> y;
    Array<real_type, 6> y;
    y[0] = param.radius;
    y[1] = 0.0;
    y[2] = tid.get() * 1.0e-6; //!< XXX use random position here
    y[3] = 0;
    y[4] = param.momentum_y;
    y[5] = param.momentum_z;

    // The rhs of the equation and a temporary array
    //    OdeArray<real_type, 6> dydx;
    //    OdeArray<real_type, 6> yout;
    Array<real_type, 6> dydx;
    Array<real_type, 6> yout;

    // Test parameters and the sub-step size
    real_type hstep       = 2.0 * constants::pi * param.radius / param.nsteps;
    real_type total_error = 0;

    for (int nr = 0; nr < param.revolutions; ++nr)
    {
        // Travel hstep for nsteps times in the field
        for (CELER_MAYBE_UNUSED int i : celeritas::range(param.nsteps))
        {
            dydx = equation(y);
            yout = rk4(hstep, y, dydx);
            //            printf("yout[0]=%g dydx[1]=%g\n",yout[0],dydx[0]);
            real_type error = rk4.error(hstep, y);
            for (int i = 0; i != 6; ++i)
                y[i] = yout[i];
            total_error += error;
        }
    }

    // Output for verification
    pos_x[tid.get()] = y[0];
    pos_z[tid.get()] = y[2];
    mom_y[tid.get()] = y[4];
    mom_z[tid.get()] = y[5];
    error[tid.get()] = total_error;
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
RK4TestOutput rk4_test(FieldTestParams test_param)
{
    // Output data for kernel
    thrust::device_vector<real_type> pos_x(test_param.nstates, 0.0);
    thrust::device_vector<real_type> pos_z(test_param.nstates, 0.0);
    thrust::device_vector<real_type> mom_y(test_param.nstates, 0.0);
    thrust::device_vector<real_type> mom_z(test_param.nstates, 0.0);
    thrust::device_vector<real_type> error(test_param.nstates, 0.0);

    // Run kernel
    celeritas::KernelParamCalculator calc_launch_params(rk4_test_kernel,
                                                        "rk4_test");
    auto params = calc_launch_params(test_param.nstates);

    rk4_test_kernel<<<params.grid_size, params.block_size>>>(
        test_param,
        raw_pointer_cast(pos_x.data()),
        raw_pointer_cast(pos_z.data()),
        raw_pointer_cast(mom_y.data()),
        raw_pointer_cast(mom_z.data()),
        raw_pointer_cast(error.data()));
    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Copy result back to CPU
    RK4TestOutput result;

    result.pos_x.resize(pos_x.size());
    thrust::copy(pos_x.begin(), pos_x.end(), result.pos_x.begin());

    result.pos_z.resize(pos_z.size());
    thrust::copy(pos_z.begin(), pos_z.end(), result.pos_z.begin());

    result.mom_y.resize(mom_y.size());
    thrust::copy(mom_y.begin(), mom_y.end(), result.mom_y.begin());

    result.mom_z.resize(mom_z.size());
    thrust::copy(mom_z.begin(), mom_z.end(), result.mom_z.begin());

    result.error.resize(error.size());
    thrust::copy(error.begin(), error.end(), result.error.begin());

    return result;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
