#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020-2022 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file Steppers.test.cu
//---------------------------------------------------------------------------//
#include "Steppers.test.hh"

#include "base/Constants.hh"
#include "base/KernelParamCalculator.device.hh"
#include "base/Range.hh"
#include "base/Types.hh"
#include "base/Units.hh"
#include "field/DormandPrinceStepper.hh"
#include "field/HelixStepper.hh"
#include "field/MagFieldEquation.hh"
#include "field/RungeKuttaStepper.hh"
#include "field/Types.hh"
#include "field/UniformMagField.hh"
#include "physics/base/Units.hh"

#include "detail/MagTestTraits.hh"

using celeritas::detail::truncation_error;
using thrust::raw_pointer_cast;
using namespace celeritas;

namespace celeritas_test
{
//---------------------------------------------------------------------------//
// HELP FUNCTIONS
//---------------------------------------------------------------------------//
template<template<class> class TStepper>
__device__ inline void gpu_stepper(celeritas_test::FieldTestParams param,
                                   real_type*                      pos_x,
                                   real_type*                      pos_z,
                                   real_type*                      mom_y,
                                   real_type*                      mom_z,
                                   real_type*                      error)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= param.nstates)
        return;

    // Construct a TStepper for testing
    UniformMagField field({0, 0, param.field_value});

    using RKTraits = detail::MagTestTraits<UniformMagField, TStepper>;
    typename RKTraits::Equation_t equation(field, units::ElementaryCharge{-1});
    typename RKTraits::Stepper_t  rk4(equation);

    // Initial state and the epected state after revolutions
    OdeState y;
    y.pos = {param.radius, 0.0, tid.get() * 1.0e-6};
    y.mom = {0.0, param.momentum_y, param.momentum_z};

    // Test parameters and the sub-step size
    real_type hstep       = 2.0 * constants::pi * param.radius / param.nsteps;
    real_type total_error = 0;

    for (auto nr : range(param.revolutions))
    {
        // Travel hstep for nsteps times in the field
        for (CELER_MAYBE_UNUSED int i : celeritas::range(param.nsteps))
        {
            StepperResult result = rk4(hstep, y);
            y                    = result.end_state;
            total_error += truncation_error(hstep, 0.001, y, result.err_state);
        }
    }
    // Output for verification
    pos_x[tid.get()] = y.pos[0];
    pos_z[tid.get()] = y.pos[2];
    mom_y[tid.get()] = y.mom[1];
    mom_z[tid.get()] = y.mom[2];
    error[tid.get()] = total_error;
}

//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
__global__ void helix_test_kernel(FieldTestParams param,
                                  real_type*      pos_x,
                                  real_type*      pos_z,
                                  real_type*      mom_y,
                                  real_type*      mom_z,
                                  real_type*      error)
{
    gpu_stepper<RungeKuttaStepper>(param, pos_x, pos_z, mom_y, mom_z, error);
}

//---------------------------------------------------------------------------//
__global__ void rk4_test_kernel(FieldTestParams param,
                                real_type*      pos_x,
                                real_type*      pos_z,
                                real_type*      mom_y,
                                real_type*      mom_z,
                                real_type*      error)
{
    gpu_stepper<RungeKuttaStepper>(param, pos_x, pos_z, mom_y, mom_z, error);
}

//---------------------------------------------------------------------------//
__global__ void dp547_test_kernel(FieldTestParams param,
                                  real_type*      pos_x,
                                  real_type*      pos_z,
                                  real_type*      mom_y,
                                  real_type*      mom_z,
                                  real_type*      error)
{
    gpu_stepper<DormandPrinceStepper>(param, pos_x, pos_z, mom_y, mom_z, error);
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run the Helix stepper on device and return results
StepperTestOutput helix_test(FieldTestParams test_param)
{
    // Output data for kernel
    thrust::device_vector<real_type> pos_x(test_param.nstates, 0.0);
    thrust::device_vector<real_type> pos_z(test_param.nstates, 0.0);
    thrust::device_vector<real_type> mom_y(test_param.nstates, 0.0);
    thrust::device_vector<real_type> mom_z(test_param.nstates, 0.0);
    thrust::device_vector<real_type> error(test_param.nstates, 0.0);

    // Run kernel
    celeritas::KernelParamCalculator calc_launch_params(helix_test_kernel,
                                                        "helix_test");
    auto params = calc_launch_params(test_param.nstates);

    helix_test_kernel<<<params.blocks_per_grid, params.threads_per_block>>>(
        test_param,
        raw_pointer_cast(pos_x.data()),
        raw_pointer_cast(pos_z.data()),
        raw_pointer_cast(mom_y.data()),
        raw_pointer_cast(mom_z.data()),
        raw_pointer_cast(error.data()));
    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());

    // Copy result back to CPU
    return copy_to_cpu(pos_x, pos_z, mom_y, mom_z, error);
}

//! Run the classical Runge-Kutta stepper on device and return results
StepperTestOutput rk4_test(FieldTestParams test_param)
{
    // Output data for kernel
    thrust::device_vector<real_type> pos_x(test_param.nstates, 0.0);
    thrust::device_vector<real_type> pos_z(test_param.nstates, 0.0);
    thrust::device_vector<real_type> mom_y(test_param.nstates, 0.0);
    thrust::device_vector<real_type> mom_z(test_param.nstates, 0.0);
    thrust::device_vector<real_type> error(test_param.nstates, 0.0);

    // Run kernel
    celeritas::KernelParamCalculator calc_launch_params(rk4_test_kernel,
                                                        "rk4_test");
    auto params = calc_launch_params(test_param.nstates);

    rk4_test_kernel<<<params.blocks_per_grid, params.threads_per_block>>>(
        test_param,
        raw_pointer_cast(pos_x.data()),
        raw_pointer_cast(pos_z.data()),
        raw_pointer_cast(mom_y.data()),
        raw_pointer_cast(mom_z.data()),
        raw_pointer_cast(error.data()));
    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());

    // Copy result back to CPU
    return copy_to_cpu(pos_x, pos_z, mom_y, mom_z, error);
}

//---------------------------------------------------------------------------//
//! Run the DormandPrince547 stepper on device and return results
StepperTestOutput dp547_test(FieldTestParams test_param)
{
    // Output data for kernel
    thrust::device_vector<real_type> pos_x(test_param.nstates, 0.0);
    thrust::device_vector<real_type> pos_z(test_param.nstates, 0.0);
    thrust::device_vector<real_type> mom_y(test_param.nstates, 0.0);
    thrust::device_vector<real_type> mom_z(test_param.nstates, 0.0);
    thrust::device_vector<real_type> error(test_param.nstates, 0.0);

    // Run kernel
    celeritas::KernelParamCalculator calc_launch_params(dp547_test_kernel,
                                                        "dp547_test");
    auto params = calc_launch_params(test_param.nstates);

    dp547_test_kernel<<<params.blocks_per_grid, params.threads_per_block>>>(
        test_param,
        raw_pointer_cast(pos_x.data()),
        raw_pointer_cast(pos_z.data()),
        raw_pointer_cast(mom_y.data()),
        raw_pointer_cast(mom_z.data()),
        raw_pointer_cast(error.data()));
    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());

    // Copy result back to CPU
    return copy_to_cpu(pos_x, pos_z, mom_y, mom_z, error);
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
