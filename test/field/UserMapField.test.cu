#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020-2022 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file UserMapField.test.cu
//---------------------------------------------------------------------------//
#include <thrust/device_vector.h>

#include "base/device_runtime_api.h"
#include "base/Constants.hh"
#include "base/KernelParamCalculator.device.hh"
#include "base/Range.hh"
#include "base/Types.hh"
#include "comm/Device.hh"

#include "UserField.test.hh"
#include "detail/CMSMapField.hh"
#include "detail/FieldMapData.hh"
#include "detail/MagFieldMap.hh"

using thrust::raw_pointer_cast;

namespace celeritas_test
{
using namespace celeritas;
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void fieldmap_test_kernel(UserFieldTestParams       param,
                                     detail::FieldMapDeviceRef field_data,
                                     real_type*                value_x,
                                     real_type*                value_y,
                                     real_type*                value_z)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= param.nsamples)
        return;

    detail::CMSMapField field(field_data);
    //    Real3 pos{tid.get()*1.5-4, tid.get()*1.5-4, tid.get()*2.5-4};
    Real3 pos{tid.get() * param.delta_r,
              tid.get() * param.delta_r,
              tid.get() * param.delta_z};

    Real3 value = field(pos);

    // Output for verification
    value_x[tid.get()] = value[0];
    value_y[tid.get()] = value[1];
    value_z[tid.get()] = value[2];
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
UserFieldTestOutput fieldmap_test(UserFieldTestParams       test_param,
                                  detail::FieldMapDeviceRef field_data)
{
    // Output data for kernel
    thrust::device_vector<real_type> value_x(test_param.nsamples, 0.0);
    thrust::device_vector<real_type> value_y(test_param.nsamples, 0.0);
    thrust::device_vector<real_type> value_z(test_param.nsamples, 0.0);

    // Run kernel
    CELER_LAUNCH_KERNEL(fieldmap_test,
                        celeritas::device().default_block_size(),
                        test_param.nsamples,
                        test_param,
                        field_data,
                        raw_pointer_cast(value_x.data()),
                        raw_pointer_cast(value_y.data()),
                        raw_pointer_cast(value_z.data()));
    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());

    // Copy result back to CPU
    UserFieldTestOutput result;

    result.value_x.resize(value_x.size());
    thrust::copy(value_x.begin(), value_x.end(), result.value_x.begin());

    result.value_y.resize(value_y.size());
    thrust::copy(value_y.begin(), value_y.end(), result.value_y.begin());

    result.value_z.resize(value_z.size());
    thrust::copy(value_z.begin(), value_z.end(), result.value_z.begin());

    return result;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
