#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file UserMapField.test.cu
//---------------------------------------------------------------------------//
#include "UserField.test.hh"
#include "detail/MagFieldMap.hh"
#include "detail/FieldMapInterface.hh"
#include "detail/CMSMapField.hh"

#include "base/KernelParamCalculator.cuda.hh"
#include <thrust/device_vector.h>

#include "base/Range.hh"
#include "base/Types.hh"
#include "base/Constants.hh"

using thrust::raw_pointer_cast;

namespace celeritas_test
{
using namespace celeritas;
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void fieldmap_test_kernel(UserFieldTestParams       param,
                                     detail::FieldMapDeviceRef group,
                                     real_type*                value_x,
                                     real_type*                value_y,
                                     real_type*                value_z)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= param.nsamples)
        return;

    detail::CMSMapField field(group);
    //    Real3 pos{tid.get()*1.5-4, tid.get()*1.5-4, tid.get()*2.5-4};
    Real3 pos{tid.get() * param.delta_r,
              tid.get() * param.delta_r,
              tid.get() * param.delta_z};

    Real3 value = field(pos);

    // Output for verification
    value_x[tid.get()] = value[0];
    value_y[tid.get()] = value[1];
    value_z[tid.get()] = value[2];
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
UserFieldTestOutput
fieldmap_test(UserFieldTestParams test_param, detail::FieldMapDeviceRef group)
{
    // Output data for kernel
    thrust::device_vector<real_type> value_x(test_param.nsamples, 0.0);
    thrust::device_vector<real_type> value_y(test_param.nsamples, 0.0);
    thrust::device_vector<real_type> value_z(test_param.nsamples, 0.0);

    // Run kernel
    celeritas::KernelParamCalculator calc_launch_params(fieldmap_test_kernel,
                                                        "fieldmap_test");
    auto params = calc_launch_params(test_param.nsamples);

    fieldmap_test_kernel<<<params.grid_size, params.block_size>>>(
        test_param,
        group,
        raw_pointer_cast(value_x.data()),
        raw_pointer_cast(value_y.data()),
        raw_pointer_cast(value_z.data()));
    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Copy result back to CPU
    UserFieldTestOutput result;

    result.value_x.resize(value_x.size());
    thrust::copy(value_x.begin(), value_x.end(), result.value_x.begin());

    result.value_y.resize(value_y.size());
    thrust::copy(value_y.begin(), value_y.end(), result.value_y.begin());

    result.value_z.resize(value_z.size());
    thrust::copy(value_z.begin(), value_z.end(), result.value_z.begin());

    return result;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
