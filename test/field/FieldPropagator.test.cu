#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file FieldPropagator.test.cu
//---------------------------------------------------------------------------//
#include "FieldTestParams.hh"
#include "FieldPropagator.test.hh"
#include "field/FieldParamsPointers.hh"

#include "base/KernelParamCalculator.cuda.hh"
#include "geometry/GeoTrackView.hh"
#include "physics/base/ParticleTrackView.hh"

#include "field/UniformMagField.hh"
#include "field/MagFieldEquation.hh"
#include "field/RungeKuttaStepper.hh"
#include "field/FieldDriver.hh"
#include "field/FieldPropagator.hh"

#include <thrust/device_vector.h>

using thrust::raw_pointer_cast;

namespace celeritas_test
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void fp_test_kernel(const int                 size,
                               const GeoParamsCRefDevice shared,
                               const GeoStateRefDevice   state,
                               const VGGTestInit*        start,
                               ParticleParamsPointers    particle_params,
                               ParticleStatePointers     particle_states,
                               FieldParamsPointers       field_params,
                               FieldTestParams           test,
                               const ParticleTrackState* init_track,
                               double*                   pos,
                               double*                   dir,
                               double*                   step)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= size)
        return;

    // Initialize GeoTrackView and ParticleTrackView
    GeoTrackView geo_track(shared, state, tid);
    geo_track = start[tid.get()];
    if (!geo_track.is_outside())
        geo_track.find_next_step();

    ParticleTrackView particle_track(particle_params, particle_states, tid);
    particle_track = init_track[tid.get()];

    // Construct the RK stepper adnd propagator in a field
    UniformMagField                   field({0, 0, test.field_value});
    MagFieldEquation<UniformMagField> eq(field, units::ElementaryCharge{-1});
    RungeKuttaStepper<UniformMagField, MagFieldEquation> rk4(eq);
    FieldDriver<UniformMagField, MagFieldEquation> driver(field_params, rk4);
    FieldPropagator<UniformMagField, MagFieldEquation> propagator(
        &geo_track, particle_track, driver);

    // Tests with input parameters of a electron in a uniform magnetic field
    double hstep = (2.0 * constants::pi * test.radius) / test.nsteps;

    real_type curved_length = 0;

    FieldPropagator<UniformMagField, MagFieldEquation>::result_type result;

    for (CELER_MAYBE_UNUSED int i : celeritas::range(test.revolutions))
    {
        for (CELER_MAYBE_UNUSED int j : celeritas::range(test.nsteps))
        {
            result = propagator(hstep);
            curved_length += result.distance;
        }
    }

    // output
    step[tid.get()] = curved_length;
    pos[tid.get()]  = geo_track.pos()[0];
    dir[tid.get()]  = geo_track.dir()[1];
}

__global__ void bc_test_kernel(const int                 size,
                               const GeoParamsCRefDevice shared,
                               const GeoStateRefDevice   state,
                               const VGGTestInit*        start,
                               ParticleParamsPointers    particle_params,
                               ParticleStatePointers     particle_states,
                               FieldParamsPointers       field_params,
                               FieldTestParams           test,
                               const ParticleTrackState* init_track,
                               double*                   pos,
                               double*                   dir,
                               double*                   step)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= size)
        return;

    // Initialize GeoTrackView and ParticleTrackView
    GeoTrackView geo_track(shared, state, tid);
    geo_track = start[tid.get()];
    if (!geo_track.is_outside())
        geo_track.find_next_step();

    ParticleTrackView particle_track(particle_params, particle_states, tid);
    particle_track = init_track[tid.get()];

    // Construct the RK stepper and propagator in a field
    UniformMagField                   field({0, 0, test.field_value});
    MagFieldEquation<UniformMagField> eq(field, units::ElementaryCharge{-1});
    RungeKuttaStepper<UniformMagField, MagFieldEquation> rk4(eq);
    FieldDriver<UniformMagField, MagFieldEquation> driver(field_params, rk4);
    FieldPropagator<UniformMagField, MagFieldEquation> propagator(
        &geo_track, particle_track, driver);

    // Tests with input parameters of a electron in a uniform magnetic field
    double hstep = (2.0 * constants::pi * test.radius) / test.nsteps;

    real_type curved_length = 0;

    constexpr int num_boundary = 16;
    int           icross       = 0;

    // clang-format off
    constexpr real_type expected_y[num_boundary]
        = { 0.5,  1.5,  2.5,  3.5,  3.5,  2.5,  1.5,  0.5,
           -0.5, -1.5, -2.5, -3.5, -3.5, -2.5, -1.5, -0.5};
    // clang-format on

    real_type delta = celeritas::numeric_limits<real_type>::max();

    FieldPropagator<UniformMagField, MagFieldEquation>::result_type result;

    for (CELER_MAYBE_UNUSED int ir : celeritas::range(test.revolutions))
    {
        for (CELER_MAYBE_UNUSED int i : celeritas::range(test.nsteps))
        {
            result = propagator(hstep);
            curved_length += result.distance;

            if (result.on_boundary)
            {
                icross++;
                int j = (icross - 1) % num_boundary;
                delta = expected_y[j] - geo_track.pos()[1];
                if (delta != 0)
                {
                    printf("Intersection Finding Failed on GPU: ");
                    printf("Expected = %f Actual = %f\n",
                           expected_y[j],
                           geo_track.pos()[1]);
                }
            }
        }
    }

    // output
    step[tid.get()] = curved_length;
    pos[tid.get()]  = geo_track.pos()[0];
    Real3 final_dir = geo_track.dir();
    normalize_direction(&final_dir);
    dir[tid.get()] = final_dir[1];
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
FPTestOutput fp_test(FPTestInput input)
{
    CELER_ASSERT(input.init_geo.size() == input.init_track.size());
    CELER_ASSERT(input.geo_params);
    CELER_ASSERT(input.geo_states);

    // Temporary device data for kernel
    thrust::device_vector<VGGTestInit>        in_geo(input.init_geo.begin(),
                                              input.init_geo.end());
    thrust::device_vector<ParticleTrackState> in_track = input.init_track;

    // Output data for kernel
    thrust::device_vector<double> step(input.init_geo.size(), -1.0);
    thrust::device_vector<double> pos(input.init_geo.size(), -1.0);
    thrust::device_vector<double> dir(input.init_geo.size(), -1.0);

    // Run kernel
    celeritas::KernelParamCalculator calc_launch_params(fp_test_kernel,
                                                        "fp_test");
    auto params = calc_launch_params(in_geo.size());

    fp_test_kernel<<<params.grid_size, params.block_size>>>(
        in_geo.size(),
        input.geo_params,
        input.geo_states,
        raw_pointer_cast(in_geo.data()),
        input.particle_params,
        input.particle_states,
        input.field_params,
        input.test,
        raw_pointer_cast(in_track.data()),
        raw_pointer_cast(pos.data()),
        raw_pointer_cast(dir.data()),
        raw_pointer_cast(step.data()));
    CELER_CUDA_CHECK_ERROR();
    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Copy result back to CPU
    FPTestOutput result;

    result.step.resize(step.size());
    thrust::copy(step.begin(), step.end(), result.step.begin());

    result.pos.resize(pos.size());
    thrust::copy(pos.begin(), pos.end(), result.pos.begin());

    result.dir.resize(dir.size());
    thrust::copy(dir.begin(), dir.end(), result.dir.begin());

    return result;
}

//! Run a boundary crossing test on device and return results

FPTestOutput bc_test(FPTestInput input)
{
    CELER_ASSERT(input.init_geo.size() == input.init_track.size());
    CELER_ASSERT(input.geo_params);
    CELER_ASSERT(input.geo_states);

    // Temporary device data for kernel
    thrust::device_vector<VGGTestInit>        in_geo(input.init_geo.begin(),
                                              input.init_geo.end());
    thrust::device_vector<ParticleTrackState> in_track = input.init_track;

    // Output data for kernel
    thrust::device_vector<double> step(input.init_geo.size(), -1.0);
    thrust::device_vector<double> pos(input.init_geo.size(), -1.0);
    thrust::device_vector<double> dir(input.init_geo.size(), -1.0);

    // Run kernel
    celeritas::KernelParamCalculator calc_launch_params(bc_test_kernel,
                                                        "bc_test");
    auto params = calc_launch_params(in_geo.size());

    bc_test_kernel<<<params.grid_size, params.block_size>>>(
        in_geo.size(),
        input.geo_params,
        input.geo_states,
        raw_pointer_cast(in_geo.data()),
        input.particle_params,
        input.particle_states,
        input.field_params,
        input.test,
        raw_pointer_cast(in_track.data()),
        raw_pointer_cast(pos.data()),
        raw_pointer_cast(dir.data()),
        raw_pointer_cast(step.data()));
    CELER_CUDA_CHECK_ERROR();
    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Copy result back to CPU
    FPTestOutput result;

    result.step.resize(step.size());
    thrust::copy(step.begin(), step.end(), result.step.begin());

    result.pos.resize(pos.size());
    thrust::copy(pos.begin(), pos.end(), result.pos.begin());

    result.dir.resize(dir.size());
    thrust::copy(dir.begin(), dir.end(), result.dir.begin());

    return result;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
