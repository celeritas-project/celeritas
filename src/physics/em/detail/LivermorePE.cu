#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file LivermorePE.cu
//---------------------------------------------------------------------------//
#include "LivermorePE.hh"

#include "base/KernelParamCalculator.cuda.hh"
#include "random/cuda/RngEngine.hh"
#include "physics/base/ModelInterface.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/PhysicsTrackView.hh"
#include "physics/base/SecondaryAllocatorView.hh"
#include "physics/material/ElementSelector.hh"
#include "physics/material/MaterialTrackView.hh"
#include "LivermorePEInteractor.hh"
#include "LivermorePEMicroXsCalculator.hh"

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Interact using the Livermore photoelectric model on applicable tracks.
 */
__global__ void livermore_pe_interact_kernel(const LivermorePEPointers   pe,
                                             const ModelInteractPointers ptrs)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= ptrs.states.size())
        return;

    SecondaryAllocatorView allocate_secondaries(ptrs.secondaries);
    ParticleTrackView particle(ptrs.params.particle, ptrs.states.particle, tid);
    MaterialTrackView material(ptrs.params.material, ptrs.states.material, tid);
    PhysicsTrackView  physics(ptrs.params.physics,
                             ptrs.states.physics,
                             particle.particle_id(),
                             material.material_id(),
                             tid);

    // This interaction only applies if the Livermore PE model was selected
    if (physics.model_id() != pe.model_id)
        return;

    RngEngine rng(ptrs.states.rng, tid);

    // Sample an element
    ElementSelector select_el(
        material.material_view(),
        LivermorePEMicroXsCalculator{pe, particle.energy()},
        material.element_scratch());
    ElementComponentId comp_id = select_el(rng);
    ElementId          el_id   = material.material_view().element_id(comp_id);

    LivermorePEInteractor interact(pe,
                                   el_id,
                                   particle,
                                   ptrs.states.direction[tid.get()],
                                   allocate_secondaries);

    ptrs.result[tid.get()] = interact(rng);
    CELER_ENSURE(ptrs.result[tid.get()]);
}

} // namespace

//---------------------------------------------------------------------------//
// LAUNCHERS
//---------------------------------------------------------------------------//
/*!
 * Launch the Livermore photoelectric interaction.
 */
void livermore_pe_interact(const LivermorePEPointers&   pe,
                           const ModelInteractPointers& model)
{
    CELER_EXPECT(pe);
    CELER_EXPECT(model);

    static const KernelParamCalculator calc_kernel_params(
        livermore_pe_interact_kernel, "livermore_pe_interact");
    auto                  params = calc_kernel_params(model.states.size());
    livermore_pe_interact_kernel<<<params.grid_size, params.block_size>>>(
        pe, model);
    CELER_CUDA_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
} // namespace detail
} // namespace celeritas
