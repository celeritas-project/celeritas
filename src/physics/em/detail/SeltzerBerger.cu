#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file SeltzerBerger.cu
//---------------------------------------------------------------------------//
#include "SeltzerBerger.hh"

#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "base/StackAllocator.hh"
#include "physics/base/ModelInterface.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/PhysicsTrackView.hh"
#include "physics/material/MaterialTrackView.hh"
#include "random/RngEngine.hh"
#include "SeltzerBergerInteractor.hh"

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Interact using the Seltzer-Berger model on applicable tracks.
 */
__global__ void seltzer_berger_interact_kernel(
    const SeltzerBergerDeviceRef&              device_pointers,
    const ModelInteractRefs<MemSpace::device>& interaction)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (!(tid < interaction.states.size()))
        return;

    ParticleTrackView particle(
        interaction.params.particle, interaction.states.particle, tid);

    // Setup for ElementView access
    MaterialTrackView material(
        interaction.params.material, interaction.states.material, tid);

    PhysicsTrackView physics(interaction.params.physics,
                             interaction.states.physics,
                             particle.particle_id(),
                             material.material_id(),
                             tid);

    // This interaction only applies if the Seltzer-Berger model was selected
    if (physics.model_id() != device_pointers.ids.model)
        return;

    // Assume only a single element in the material, for now
    MaterialView    material_view = material.material_view();
    CELER_ASSERT(material_view.num_elements() == 1);
    const ElementId element_id{0};

    CutoffView cutoffs(interaction.params.cutoffs, material.material_id());
    StackAllocator<Secondary> allocate_secondaries(
        interaction.states.secondaries);
    SeltzerBergerInteractor interact(device_pointers,
                                     particle,
                                     interaction.states.direction[tid],
                                     cutoffs,
                                     allocate_secondaries,
                                     material_view,
                                     element_id);

    RngEngine rng(interaction.states.rng, tid);
    interaction.states.interactions[tid] = interact(rng);
    CELER_ENSURE(interaction.states.interactions[tid]);
}

} // namespace

//---------------------------------------------------------------------------//
// LAUNCHERS
//---------------------------------------------------------------------------//
/*!
 * Launch the Seltzer-Berger interaction.
 */
void seltzer_berger_interact(
    const SeltzerBergerDeviceRef&              device_pointers,
    const ModelInteractRefs<MemSpace::device>& interaction)
{
    CELER_EXPECT(device_pointers);
    CELER_EXPECT(interaction);

    static const KernelParamCalculator calc_kernel_params(
        seltzer_berger_interact_kernel, "seltzer_berger_interact");
    auto params = calc_kernel_params(interaction.states.size());
    seltzer_berger_interact_kernel<<<params.grid_size, params.block_size>>>(
        device_pointers, interaction);
    CELER_CUDA_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
} // namespace detail
} // namespace celeritas
