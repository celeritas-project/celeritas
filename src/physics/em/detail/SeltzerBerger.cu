#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file SeltzerBerger.cu
//---------------------------------------------------------------------------//
#include "SeltzerBerger.hh"

#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "random/RngEngine.hh"
#include "physics/base/ModelInterface.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/PhysicsTrackView.hh"
#include "base/StackAllocator.hh"
#include "physics/material/MaterialTrackView.hh"
#include "SeltzerBergerInteractor.hh"

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Interact using the Seltzer-Berger model on applicable tracks.
 */
__global__ void seltzer_berger_interact_kernel(
    const SeltzerBergerDeviceRef&              device_pointers,
    const ModelInteractRefs<MemSpace::device>& interaction)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (!(tid < interaction.states.size()))
        return;

    StackAllocator<Secondary> allocate_secondaries(
        interaction.states.secondaries);
    ParticleTrackView particle(
        interaction.params.particle, interaction.states.particle, tid);

    // Setup for ElementView access
    MaterialTrackView material(
        interaction.params.material, interaction.states.material, tid);
    // Cache the associated MaterialView as function calls to MaterialTrackView
    // are expensive
    MaterialView material_view = material.material_view();

    PhysicsTrackView physics(interaction.params.physics,
                             interaction.states.physics,
                             particle.particle_id(),
                             material.material_id(),
                             tid);

    // This interaction only applies if the Seltzer-Berger model was selected
    if (physics.model_id() != device_pointers.ids.model)
        return;

    CutoffView cutoffs(interaction.params.cutoffs, material.material_id());

    // Assume only a single element in the material, for now
    CELER_ASSERT(material_view.num_elements() == 1);
    SeltzerBergerInteractor interact(device_pointers,
                                     particle,
                                     interaction.states.direction[tid],
                                     cutoffs,
                                     allocate_secondaries,
                                     material_view);

    RngEngine rng(interaction.states.rng, tid);
    interaction.states.interactions[tid] = interact(rng);
    CELER_ENSURE(interaction.states.interactions[tid]);
}

} // namespace

//---------------------------------------------------------------------------//
// LAUNCHERS
//---------------------------------------------------------------------------//
/*!
 * Launch the Seltzer-Berger interaction.
 */
void seltzer_berger_interact(
    const SeltzerBergerDeviceRef&              device_pointers,
    const ModelInteractRefs<MemSpace::device>& interaction)
{
    CELER_EXPECT(device_pointers);
    CELER_EXPECT(interaction);

    static const KernelParamCalculator calc_kernel_params(
        seltzer_berger_interact_kernel, "seltzer_berger_interact");
    auto params = calc_kernel_params(interaction.states.size());
    seltzer_berger_interact_kernel<<<params.grid_size, params.block_size>>>(
        device_pointers, interaction);
    CELER_CUDA_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
} // namespace detail
} // namespace celeritas
