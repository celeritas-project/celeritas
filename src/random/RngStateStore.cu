#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file RngStateStore.cu
//---------------------------------------------------------------------------//
#include "RngStateStore.hh"

#include <vector>
#include <thrust/device_vector.h>
#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "RngEngine.cuh"

namespace celeritas
{
//---------------------------------------------------------------------------//
struct RngStateContainerPimpl
{
    thrust::device_vector<RngState> rng;
};

//---------------------------------------------------------------------------//
/*!
 * Initialize the RNG states on device from seeds randomly generated on host.
 */
__global__ void initialize_states(RngStatePointers view, seed_type* seeds)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() < view.size)
    {
        RngEngine rng(view, tid);
        rng.initialize_state(seeds[tid.get()]);
    }
}

//---------------------------------------------------------------------------//
/*!
 * Construct with the number of RNG states.
 */
RngStateStore::RngStateStore(ssize_type size, seed_type host_seed)
    : host_rng_(host_seed)
{
    this->resize(size);
}

//---------------------------------------------------------------------------//
// Default constructor/destructor/move
RngStateStore::RngStateStore()                = default;
RngStateStore::~RngStateStore()               = default;
RngStateStore::RngStateStore(RngStateStore&&) = default;
RngStateStore& RngStateStore::operator=(RngStateStore&&) = default;

//---------------------------------------------------------------------------//
/*!
 * Resize the RNG state vector, initializing new states if the number requested
 * is larger than the current size.
 */
void RngStateStore::resize(ssize_type size)
{
    int num_states     = this->size();
    int num_new_states = size - num_states;

    // Allocate and copy data to device
    if (this->size() == 0)
    {
        data_ = std::make_unique<RngStateContainerPimpl>();
    }
    data_->rng.resize(size);
    size_ = size;

    if (num_new_states > 0)
    {
        // Create seeds on host
        std::vector<seed_type> host_seeds(num_new_states);
        for (auto& seed : host_seeds)
            seed = sample_uniform_int_(host_rng_);

        // Copy seeds to device
        thrust::device_vector<seed_type> seeds = host_seeds;

        // Create a view of new states to initialize
        RngStatePointers view;
        view.size = num_new_states;
        view.rng  = thrust::raw_pointer_cast(data_->rng.data()) + num_states;

        // Launch kernel to build RNG states on device
        celeritas::KernelParamCalculator calc_launch_params;
        auto params = calc_launch_params(num_new_states);
        initialize_states<<<params.grid_size, params.block_size>>>(
            view, thrust::raw_pointer_cast(seeds.data()));
    }
    ENSURE(data_->rng.size() == size);
}

//---------------------------------------------------------------------------//
/*!
 * Return a view to on-device memory
 */
RngStatePointers RngStateStore::device_pointers() const
{
    REQUIRE(data_);

    RngStatePointers view;
    view.size = data_->rng.size();
    view.rng  = thrust::raw_pointer_cast(data_->rng.data());

    return view;
}

//---------------------------------------------------------------------------//
} // namespace celeritas
