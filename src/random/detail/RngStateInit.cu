#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020-2022 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file RngStateInit.cu
//---------------------------------------------------------------------------//
#include "RngStateInit.hh"

#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "random/RngEngine.hh"

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Initialize the RNG states on device from seeds randomly generated on host.
 */
__global__ void rng_state_init_kernel(
    RngStateData<Ownership::reference, MemSpace::device> const      state,
    RngInitData<Ownership::const_reference, MemSpace::device> const init)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() < state.size())
    {
        RngEngine rng(state, tid);
        rng = init.seeds[tid];
    }
}

//---------------------------------------------------------------------------//
} // namespace

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 * Initialize the RNG states on device from seeds randomly generated on host.
 */
void rng_state_init(
    const RngStateData<Ownership::reference, MemSpace::device>&      rng,
    const RngInitData<Ownership::const_reference, MemSpace::device>& seeds)
{
    CELER_EXPECT(rng.size() == seeds.size());

    // Launch kernel to build RNG states on device
    static const celeritas::KernelParamCalculator calc_launch_params(
        rng_state_init_kernel, "rng_state_init");
    auto params = calc_launch_params(seeds.size());
    rng_state_init_kernel<<<params.grid_size, params.block_size>>>(rng, seeds);
    CELER_CUDA_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
} // namespace detail
} // namespace celeritas
