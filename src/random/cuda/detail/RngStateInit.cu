#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file RngStateInit.cu
//---------------------------------------------------------------------------//
#include "RngStateInit.hh"

#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "random/cuda/RngEngine.hh"

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Initialize the RNG states on device from seeds randomly generated on host.
 */
__global__ void rng_init_kernel(const RngStatePointers           state,
                                const RngSeed::value_type* const seeds)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() < state.size())
    {
        RngEngine rng(state, tid);
        rng = RngEngine::Initializer_t{seeds[tid.get()]};
    }
}
//---------------------------------------------------------------------------//
} // namespace

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 * Initialize the RNG states on device from seeds randomly generated on host.
 */
void rng_state_init_device(const RngStatePointers&         device_ptrs,
                           Span<const RngSeed::value_type> device_seeds)
{
    CELER_EXPECT(device_ptrs.size() == device_seeds.size());

    // Launch kernel to build RNG states on device
    celeritas::KernelParamCalculator calc_launch_params;
    auto params = calc_launch_params(device_seeds.size());
    rng_init_kernel<<<params.grid_size, params.block_size>>>(
        device_ptrs, device_seeds.data());
    CELER_CUDA_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
} // namespace detail
} // namespace celeritas
