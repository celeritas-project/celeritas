#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/track/InitializeTracksAction.cu
//---------------------------------------------------------------------------//
#include "InitializeTracksAction.hh"

#include "corecel/device_runtime_api.h"
#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"

#include "detail/InitTracksLauncher.hh"

namespace celeritas
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
__global__ void init_tracks_kernel(detail::InitTracksLauncher launch)
{
    launch(KernelParamCalculator::thread_id());
}
//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Launch a kernel to initialize tracks.
 */
void InitializeTracksAction::launch_impl(CoreParams const& params,
                                         CoreStateDevice& state,
                                         size_type num_new_tracks) const
{
    CELER_LAUNCH_KERNEL(
        init_tracks,
        celeritas::device().default_block_size(),
        num_new_tracks,
        detail::InitTracksLauncher{params.ptr<MemSpace::device>(),
                                   state.ptr(),
                                   num_new_tracks,
                                   state.counters()});
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
