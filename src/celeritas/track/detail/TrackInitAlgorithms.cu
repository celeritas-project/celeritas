//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020-2024 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/track/detail/TrackInitAlgorithms.cu
//---------------------------------------------------------------------------//
#include "TrackInitAlgorithms.hh"

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/remove.h>
#include <thrust/scan.h>
#include <thrust/sort.h>

#include "corecel/Macros.hh"
#include "corecel/data/ObserverPtr.device.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/ScopedProfiling.hh"
#include "corecel/sys/Stream.hh"
#include "corecel/sys/Thrust.device.hh"

#include "Utils.hh"

namespace celeritas
{
namespace detail
{
//---------------------------------------------------------------------------//
/*!
 * Remove all elements in the vacancy vector that were flagged as active
 * tracks.
 */
size_type remove_if_alive(
    StateCollection<TrackSlotId, Ownership::reference, MemSpace::device> const&
        vacancies,
    StreamId stream_id)
{
    ScopedProfiling profile_this{"remove-if-alive"};
    auto start = device_pointer_cast(vacancies.data());
    auto end = thrust::remove_if(thrust_execute_on(stream_id),
                                 start,
                                 start + vacancies.size(),
                                 IsEqual{occupied()});
    CELER_DEVICE_CHECK_ERROR();

    // New size of the vacancy vector
    return end - start;
}

//---------------------------------------------------------------------------//
/*!
 * Do an exclusive scan of the number of secondaries produced by each track.
 *
 * For an input array x, this calculates the exclusive prefix sum y of the
 * array elements, i.e., \f$ y_i = \sum_{j=0}^{i-1} x_j \f$,
 * where \f$ y_0 = 0 \f$, and stores the result in the input array.
 *
 * The return value is the sum of all elements in the input array.
 */
size_type exclusive_scan_counts(
    StateCollection<size_type, Ownership::reference, MemSpace::device> const&
        counts,
    StreamId stream_id)
{
    ScopedProfiling profile_this{"exclusive-scan-conts"};
    // Exclusive scan:
    auto data = device_pointer_cast(counts.data());
    auto stop = thrust::exclusive_scan(thrust_execute_on(stream_id),
                                       data,
                                       data + counts.size(),
                                       data,
                                       size_type(0));
    CELER_DEVICE_CHECK_ERROR();

    // Copy the last element (accumulated total) back to host
    return *(stop - 1);
}

//---------------------------------------------------------------------------//
/*!
 * Sort the tracks that will be initialized in this step by charged/neutral.
 *
 * \note This implementaion uses sort rather than partition to avoid the
 * blocking device-to-host copy.
 */
void partition_initializers(
    CoreParams const& params,
    Collection<TrackInitializer, Ownership::reference, MemSpace::device> const&
        init,
    CoreStateCounters const& counters,
    size_type count,
    StreamId stream_id)
{
    ScopedProfiling profile_this{"partition-initializers"};
    auto end = device_pointer_cast(init.data()) + counters.num_initializers;
    auto start = end - count;
    thrust::stable_sort(thrust_execute_on(stream_id),
                        start,
                        end,
                        IsNeutralFirst{params.ptr<MemSpace::native>()});
    CELER_DEVICE_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
}  // namespace detail
}  // namespace celeritas
