#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/track/detail/TrackSortUtils.cu
//---------------------------------------------------------------------------//
#include "TrackSortUtils.hh"

#include <random>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/partition.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/shuffle.h>
#include <thrust/sort.h>

#include "corecel/Macros.hh"
#include "corecel/data/Collection.hh"
#include "corecel/data/Copier.hh"
#include "corecel/data/ObserverPtr.device.hh"
#include "corecel/data/ObserverPtr.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//

template<class T>
using StateItems
    = celeritas::StateCollection<T, Ownership::reference, MemSpace::device>;

template<class T>
using ThreadItems
    = Collection<T, Ownership::reference, MemSpace::device, ThreadId>;

using TrackSlots = ThreadItems<TrackSlotId::size_type>;

//---------------------------------------------------------------------------//

template<class F>
void partition_impl(TrackSlots const& track_slots, F&& func)
{
    auto start = device_pointer_cast(track_slots.data());
    thrust::partition(thrust::device,
                      start,
                      start + track_slots.size(),
                      std::forward<F>(func));
    CELER_DEVICE_CHECK_ERROR();
}

//---------------------------------------------------------------------------//

template<class F>
void sort_impl(TrackSlots const& track_slots, F&& func)
{
    auto start = device_pointer_cast(track_slots.data());
    thrust::sort(thrust::device,
                 start,
                 start + track_slots.size(),
                 std::forward<F>(func));
    CELER_DEVICE_CHECK_ERROR();
}

// PRE: action_accessor is sorted, i.e. i <= j ==> action_accessor(i) <=
// action_accessor(j)
template<class F>
CELER_FUNCTION void tracks_per_action_impl(Span<ThreadId> offsets,
                                           size_type size,
                                           F&& action_accessor)
{
    ThreadId tid = celeritas::KernelParamCalculator::thread_id();

    if ((tid < size) && tid.get() != 0)
    {
        ActionId current_action = action_accessor(tid);
        ActionId previous_action = action_accessor(tid - 1);
        if (current_action && current_action != previous_action)
        {
            offsets[current_action.get()] = tid;
        }
    }
    // needed if the first action range has only one element
    if (ActionId first; tid.get() == 0 && (first = action_accessor(tid)))
    {
        offsets[first.get()] = tid;
    }
}

__global__ void tracks_per_action_kernel(DeviceRef<CoreStateData> const states,
                                         Span<ThreadId> offsets,
                                         size_type size,
                                         TrackOrder order)
{
    switch (order)
    {
        case TrackOrder::sort_along_step_action:
            return tracks_per_action_impl(
                offsets,
                size,
                along_step_action_accessor{states.sim.along_step_action.data(),
                                           states.track_slots.data()});
        case TrackOrder::sort_step_limit_action:
            return tracks_per_action_impl(
                offsets,
                size,
                step_limit_action_accessor{states.sim.step_limit.data(),
                                           states.track_slots.data()});
        default:
            CELER_ASSERT_UNREACHABLE();
    }
}

//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Initialize default threads to track_slots mapping, track_slots[i] = i.
 *
 * TODO: move to global/detail
 */
template<>
void fill_track_slots<MemSpace::device>(Span<TrackSlotId::size_type> track_slots)
{
    thrust::sequence(
        thrust::device_pointer_cast(track_slots.data()),
        thrust::device_pointer_cast(track_slots.data() + track_slots.size()),
        0);
    CELER_DEVICE_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
/*!
 * Shuffle track slots.
 *
 * TODO: move to global/detail
 */
template<>
void shuffle_track_slots<MemSpace::device>(
    Span<TrackSlotId::size_type> track_slots)
{
    using result_type = thrust::default_random_engine::result_type;
    thrust::default_random_engine g{
        static_cast<result_type>(track_slots.size())};
    auto start = thrust::device_pointer_cast(track_slots.data());
    thrust::shuffle(thrust::device, start, start + track_slots.size(), g);
    CELER_DEVICE_CHECK_ERROR();
}

//---------------------------------------------------------------------------//
/*!
 * Sort or partition tracks.
 */
void sort_tracks(DeviceRef<CoreStateData> const& states, TrackOrder order)
{
    switch (order)
    {
        case TrackOrder::partition_status:
            return partition_impl(states.track_slots,
                                  alive_predicate{states.sim.status.data()});
        case TrackOrder::sort_along_step_action:
            sort_impl(
                states.track_slots,
                along_action_comparator{states.sim.along_step_action.data()});
            return;
        case TrackOrder::sort_step_limit_action:
            sort_impl(states.track_slots,
                      step_limit_comparator{states.sim.step_limit.data()});
            return;
        default:
            CELER_ASSERT_UNREACHABLE();
    }
}

//---------------------------------------------------------------------------//
/*!
 * Count tracks associated to each action that was used to sort them, specified
 * by order. Result is written in the output parameter offsets which sould be
 * of size num_actions + 1.
 */
template<>
void count_tracks_per_action<MemSpace::device>(
    DeviceRef<CoreStateData> const& states,
    Span<ThreadId> offsets,
    Collection<ThreadId, Ownership::value, MemSpace::host, ActionId>& out,
    TrackOrder order)
{
    switch (order)
    {
        case TrackOrder::sort_along_step_action:
        case TrackOrder::sort_step_limit_action: {
            // dispatch in the kernel since CELER_LAUNCH_KERNEL doesn't work
            // with templated kernels
            auto start = device_pointer_cast(make_observer(offsets.data()));
            thrust::fill(start, start + offsets.size(), ThreadId{});
            CELER_DEVICE_CHECK_ERROR();

            CELER_LAUNCH_KERNEL(tracks_per_action,
                                celeritas::device().default_block_size(),
                                states.size(),
                                states,
                                offsets,
                                states.size(),
                                order);

            Span<ThreadId> sout = out[AllItems<ThreadId, MemSpace::host>{}];
            Copier<ThreadId, MemSpace::host> copy_to_host{sout};
            copy_to_host(MemSpace::device, offsets);

            sout.back() = ThreadId{states.size()};

            // in case some actions were not found, have them "start" at the
            // next action offset.
            for (auto thread_id = sout.end() - 2; thread_id >= sout.begin();
                 --thread_id)
            {
                if (*thread_id == ThreadId{})
                {
                    *thread_id = *(thread_id + 1);
                }
            }
            return;
        }
        default:
            return;
    }
}

//---------------------------------------------------------------------------//
}  // namespace detail
}  // namespace celeritas
