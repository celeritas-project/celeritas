//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2024 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/track/StatusChecker.cu
//---------------------------------------------------------------------------//
#include "StatusChecker.hh"

#include "corecel/Assert.hh"
#include "corecel/sys/ScopedProfiling.hh"
#include "celeritas/global/ActionLauncher.device.hh"
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"
#include "celeritas/global/TrackExecutor.hh"

#include "detail/StatusCheckExecutor.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//

//---------------------------------------------------------------------------//
}  // namespace celeritas
