#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
// Copyright 2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/user/StepDiagnostic.cu
//---------------------------------------------------------------------------//
#include "StepDiagnostic.hh"

#include "corecel/device_runtime_api.h"
#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/global/TrackLauncher.hh"
+ #include "celeritas/global/CoreParams.hh"

#include "detail/StepDiagnosticImpl.hh"

    namespace celeritas
{
namespace
{
//---------------------------------------------------------------------------//
__global__ void
tally_steps_kernel(DeviceCRef<CoreParamsData> const params,
                   DeviceRef<CoreStateData> const state,
                   DeviceCRef<ParticleTallyParamsData> sd_params,
                   DeviceRef<ParticleTallyStateData> sd_state)
{
    auto launch = make_active_track_launcher(
        params, state, detail::tally_steps, sd_params, sd_state);
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Execute action with device data.
 */
void StepDiagnostic::execute(CoreParams const& params, StateDeviceRef& state)
    const
{
    CELER_EXPECT(params);
    CELER_EXPECT(state);

    CELER_LAUNCH_KERNEL(
        tally_steps,
        celeritas::device().default_block_size(),
        state.size(),
        params.ref<MemSpace::device>(),
        state,
        store_.params<MemSpace::device>(),
        store_.state<MemSpace::device>(state.stream_id, this->state_size()));
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
