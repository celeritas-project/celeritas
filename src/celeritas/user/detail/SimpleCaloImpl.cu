#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2023-2024 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/user/detail/SimpleCaloImpl.cu
//---------------------------------------------------------------------------//
#include "SimpleCaloImpl.hh"

#include "corecel/device_runtime_api.hh"

#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "corecel/sys/Stream.hh"

#include "SimpleCaloExecutor.hh"  // IWYU pragma: associated

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Accumulate energy deposition on device.
 */
__global__ void simple_calo_accum_kernel(DeviceRef<StepStateData> const step,
                                         DeviceRef<SimpleCaloStateData> calo)
{
    auto tid = KernelParamCalculator::thread_id();
    if (!(tid < step.size()))
        return;

    SimpleCaloExecutor execute{step, calo};
    execute(tid);
}

//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 * Accumulate energy deposition on device.
 */
void simple_calo_accum(DeviceRef<StepStateData> const& step,
                       DeviceRef<SimpleCaloStateData>& calo)
{
    CELER_EXPECT(step && calo);
    CELER_LAUNCH_KERNEL(simple_calo_accum,
                        step.size(),
                        celeritas::device().stream(step.stream_id).get(),
                        step,
                        calo);
}

//---------------------------------------------------------------------------//
}  // namespace detail
}  // namespace celeritas
