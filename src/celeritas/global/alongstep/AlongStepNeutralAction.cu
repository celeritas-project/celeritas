#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2022-2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/global/alongstep/AlongStepNeutralAction.cu
//---------------------------------------------------------------------------//
#include "AlongStepNeutralAction.hh"

#include "corecel/device_runtime_api.h"
#include "corecel/Assert.hh"
#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"
#include "celeritas/global/TrackLauncher.hh"

#include "detail/AlongStepNeutral.hh"

namespace celeritas
{
namespace
{
//---------------------------------------------------------------------------//
__global__ void
along_step_neutral_kernel(CRefPtr<CoreParamsData, MemSpace::device> const params,
                          RefPtr<CoreStateData, MemSpace::device> const state,
                          ActionId const along_step_id)
{
    auto launch = make_along_step_track_launcher(
        *params, *state, along_step_id, detail::along_step_neutral);
    launch(KernelParamCalculator::thread_id());
}
//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Launch the along-step action on device.
 */
void AlongStepNeutralAction::execute(CoreParams const& params,
                                     CoreStateDevice& state) const
{
    CELER_LAUNCH_KERNEL(along_step_neutral,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id());
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
