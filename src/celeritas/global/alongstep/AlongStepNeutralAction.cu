#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2022-2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/global/alongstep/AlongStepNeutralAction.cu
//---------------------------------------------------------------------------//
#include "AlongStepNeutralAction.hh"

#include "corecel/device_runtime_api.h"
#include "corecel/Assert.hh"
#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "corecel/sys/Stream.hh"
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"
#include "celeritas/global/TrackExecutor.hh"

#include "detail/AlongStepNeutral.hh"

namespace celeritas
{
namespace
{
//---------------------------------------------------------------------------//
__global__ void
along_step_neutral_kernel(CRefPtr<CoreParamsData, MemSpace::device> const params,
                          RefPtr<CoreStateData, MemSpace::device> const state,
                          ActionId const along_step_id)
{
    auto execute = make_along_step_track_executor(
        params, state, along_step_id, detail::along_step_neutral);
    execute(KernelParamCalculator::thread_id());
}
//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Launch the along-step action on device.
 */
void AlongStepNeutralAction::execute(CoreParams const& params,
                                     CoreStateDevice& state) const
{
    CELER_LAUNCH_KERNEL(along_step_neutral,
                        celeritas::device().default_block_size(),
                        state.size(),
                        celeritas::device().stream(state.stream_id()).get(),
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id());
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
