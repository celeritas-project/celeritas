#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2022-2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/global/alongstep/AlongStepNeutralAction.cu
//---------------------------------------------------------------------------//
#include "AlongStepNeutralAction.hh"

#include "corecel/device_runtime_api.h"
#include "corecel/Assert.hh"
#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/TrackLauncher.hh"

#include "detail/AlongStepNeutral.hh"

namespace celeritas
{
namespace
{
//---------------------------------------------------------------------------//
__global__ void
along_step_neutral_kernel(DeviceCRef<CoreParamsData> const params,
                          DeviceRef<CoreStateData> const state)
{
    auto launch = make_active_track_launcher(
        params, state, detail::along_step_neutral);
    launch(KernelParamCalculator::thread_id());
}
//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Launch the along-step action on device.
 */
void AlongStepNeutralAction::execute(CoreParams const& params,
                                     CoreStateDevice& state) const
{
    CELER_LAUNCH_KERNEL(along_step_neutral,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ref<MemSpace::native>(),
                        state);
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
