#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2022-2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/global/alongstep/AlongStepNeutralAction.cu
//---------------------------------------------------------------------------//
#include "AlongStepNeutralAction.hh"

#include "corecel/device_runtime_api.h"
#include "corecel/Assert.hh"
#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"

#include "AlongStepLauncher.hh"
#include "detail/AlongStepNeutral.hh"

namespace celeritas
{
namespace
{
//---------------------------------------------------------------------------//
__global__ void
along_step_neutral_kernel(DeviceCRef<CoreParamsData> const params,
                          DeviceRef<CoreStateData> const state)
{
    auto tid = KernelParamCalculator::thread_id();
    if (!(tid < state.size()))
        return;

    auto launch = make_along_step_launcher(
        params, state, NoData{}, NoData{}, NoData{}, detail::along_step_neutral);
    launch(tid);
}
//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Launch the along-step action on device.
 */
void AlongStepNeutralAction::execute(ParamsDeviceCRef const& params,
                                     StateDeviceRef& state) const
{
    CELER_EXPECT(params && state);
    CELER_LAUNCH_KERNEL(along_step_neutral,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params,
                        state);
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
