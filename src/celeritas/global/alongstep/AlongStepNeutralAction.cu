//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2022-2024 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/global/alongstep/AlongStepNeutralAction.cu
//---------------------------------------------------------------------------//
#include "AlongStepNeutralAction.hh"

#include "celeritas/global/ActionLauncher.device.hh"
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"
#include "celeritas/global/TrackExecutor.hh"

#include "detail/AlongStepNeutralImpl.hh"
#include "detail/LinearPropagatorFactory.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//
/*!
 * Launch the along-step action on device.
 */
void AlongStepNeutralAction::step(CoreParams const& params,
                                  CoreStateDevice& state) const
{
    auto execute = make_along_step_track_executor(
        params.ptr<MemSpace::native>(),
        state.ptr(),
        this->action_id(),
        AlongStep{detail::NoMsc{},
                  detail::LinearPropagatorFactory{},
                  detail::NoELoss{}});
    static ActionLauncher<decltype(execute)> const launch_kernel(*this);
    launch_kernel(*this, params, state, execute);
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
