#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2022-2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/global/alongstep/AlongStepGeneralLinearAction.cu
//---------------------------------------------------------------------------//
#include "AlongStepGeneralLinearAction.hh"

#include "corecel/device_runtime_api.h"
#include "corecel/Assert.hh"
#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/em/FluctuationParams.hh"
#include "celeritas/em/data/FluctuationData.hh"
#include "celeritas/em/data/UrbanMscData.hh"
#include "celeritas/em/msc/UrbanMsc.hh"  // IWYU pragma: associated
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"
#include "celeritas/global/KernelLaunchUtils.hh"
#include "celeritas/global/TrackLauncher.hh"

#include "detail/AlongStepImpl.hh"
#include "detail/AlongStepNeutral.hh"
#include "detail/FluctELoss.hh"

namespace celeritas
{
namespace
{
//---------------------------------------------------------------------------//
__global__ void along_step_apply_msc_step_limit_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id,
    ThreadId const offset,
    DeviceCRef<UrbanMscData> const msc_data)
{
    auto launch = make_along_step_track_launcher(
        *params,
        *state,
        along_step_id,
        detail::apply_msc_step_limit<UrbanMsc>,
        UrbanMsc{msc_data});
    launch(KernelParamCalculator::thread_id() + offset.get());
}

//---------------------------------------------------------------------------//
__global__ void along_step_apply_linear_propagation_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id,
    ThreadId const offset)
{
    auto launch
        = make_along_step_track_launcher(*params,
                                         *state,
                                         along_step_id,
                                         detail::ApplyPropagation{},
                                         detail::LinearPropagatorFactory{});
    launch(KernelParamCalculator::thread_id() + offset.get());
}

//---------------------------------------------------------------------------//
__global__ void along_step_apply_msc_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id,
    ThreadId const offset,
    DeviceCRef<UrbanMscData> const msc_data)
{
    auto launch = make_along_step_track_launcher(*params,
                                                 *state,
                                                 along_step_id,
                                                 detail::apply_msc<UrbanMsc>,
                                                 UrbanMsc{msc_data});
    launch(KernelParamCalculator::thread_id() + offset.get());
}

//---------------------------------------------------------------------------//
__global__ void along_step_update_time_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id,
    ThreadId const offset)
{
    auto launch = make_along_step_track_launcher(
        *params, *state, along_step_id, detail::update_time);
    launch(KernelParamCalculator::thread_id() + offset.get());
}

//---------------------------------------------------------------------------//
__global__ void along_step_apply_fluct_eloss_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id,
    ThreadId const offset,
    NativeCRef<FluctuationData> const fluct)
{
    using detail::FluctELoss;

    auto launch
        = make_along_step_track_launcher(*params,
                                         *state,
                                         along_step_id,
                                         detail::apply_eloss<FluctELoss>,
                                         FluctELoss{fluct});
    launch(KernelParamCalculator::thread_id() + offset.get());
}

//---------------------------------------------------------------------------//
__global__ void along_step_update_track_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id,
    ThreadId const offset)
{
    auto launch = make_along_step_track_launcher(
        *params, *state, along_step_id, detail::update_track);
    launch(KernelParamCalculator::thread_id() + offset.get());
}

//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Launch the along-step action on device.
 */
void AlongStepGeneralLinearAction::execute(CoreParams const& params,
                                           CoreStateDevice& state) const
{
    KernelLaunchParams kernel_params = compute_launch_params(
        this->action_id(), params, state, TrackOrder::sort_along_step_action);
    CELER_LAUNCH_KERNEL(along_step_apply_msc_step_limit,
                        celeritas::device().default_block_size(),
                        kernel_params.num_threads,
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id(),
                        kernel_params.threads_offset,
                        device_data_.msc);
    CELER_LAUNCH_KERNEL(along_step_apply_linear_propagation,
                        celeritas::device().default_block_size(),
                        kernel_params.num_threads,
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id(),
                        kernel_params.threads_offset);
    CELER_LAUNCH_KERNEL(along_step_apply_msc,
                        celeritas::device().default_block_size(),
                        kernel_params.num_threads,
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id(),
                        kernel_params.threads_offset,
                        device_data_.msc);
    CELER_LAUNCH_KERNEL(along_step_update_time,
                        celeritas::device().default_block_size(),
                        kernel_params.num_threads,
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id(),
                        kernel_params.threads_offset);
    CELER_LAUNCH_KERNEL(along_step_apply_fluct_eloss,
                        celeritas::device().default_block_size(),
                        kernel_params.num_threads,
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id(),
                        kernel_params.threads_offset,
                        device_data_.fluct);
    CELER_LAUNCH_KERNEL(along_step_update_track,
                        celeritas::device().default_block_size(),
                        kernel_params.num_threads,
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id(),
                        kernel_params.threads_offset);
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
