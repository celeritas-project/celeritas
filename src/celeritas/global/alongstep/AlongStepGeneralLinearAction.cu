#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2022-2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/global/alongstep/AlongStepGeneralLinearAction.cu
//---------------------------------------------------------------------------//
#include "AlongStepGeneralLinearAction.hh"

#include "corecel/device_runtime_api.h"
#include "corecel/Assert.hh"
#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/em/FluctuationParams.hh"
#include "celeritas/em/data/FluctuationData.hh"
#include "celeritas/em/data/UrbanMscData.hh"
#include "celeritas/em/msc/UrbanMsc.hh"  // IWYU pragma: associated
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"
#include "celeritas/global/TrackLauncher.hh"

#include "detail/AlongStepImpl.hh"
#include "detail/AlongStepNeutral.hh"
#include "detail/FluctELoss.hh"

namespace celeritas
{
namespace
{
//---------------------------------------------------------------------------//
__global__ void along_step_apply_msc_step_limit_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id,
    DeviceCRef<UrbanMscData> const msc_data)
{
    auto launch = make_along_step_track_launcher(
        *params,
        *state,
        along_step_id,
        detail::apply_msc_step_limit<UrbanMsc>,
        UrbanMsc{msc_data});
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
__global__ void along_step_apply_linear_propagation_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id)
{
    auto launch
        = make_along_step_track_launcher(*params,
                                         *state,
                                         along_step_id,
                                         detail::ApplyPropagation{},
                                         detail::LinearPropagatorFactory{});
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
__global__ void along_step_apply_msc_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id,
    DeviceCRef<UrbanMscData> const msc_data)
{
    auto launch = make_along_step_track_launcher(*params,
                                                 *state,
                                                 along_step_id,
                                                 detail::apply_msc<UrbanMsc>,
                                                 UrbanMsc{msc_data});
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
__global__ void along_step_update_time_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id)
{
    auto launch = make_along_step_track_launcher(
        *params, *state, along_step_id, detail::update_time);
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
__global__ void along_step_apply_fluct_eloss_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id,
    NativeCRef<FluctuationData> const fluct)
{
    using detail::FluctELoss;

    auto launch
        = make_along_step_track_launcher(*params,
                                         *state,
                                         along_step_id,
                                         detail::apply_eloss<FluctELoss>,
                                         FluctELoss{fluct});
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
__global__ void along_step_update_track_kernel(
    CRefPtr<CoreParamsData, MemSpace::device> const params,
    RefPtr<CoreStateData, MemSpace::device> const state,
    ActionId const along_step_id)
{
    auto launch = make_along_step_track_launcher(
        *params, *state, along_step_id, detail::update_track);
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Launch the along-step action on device.
 */
void AlongStepGeneralLinearAction::execute(CoreParams const& params,
                                           CoreStateDevice& state) const
{
    CELER_LAUNCH_KERNEL(along_step_apply_msc_step_limit,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id(),
                        device_data_.msc);
    CELER_LAUNCH_KERNEL(along_step_apply_linear_propagation,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id());
    CELER_LAUNCH_KERNEL(along_step_apply_msc,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id(),
                        device_data_.msc);
    CELER_LAUNCH_KERNEL(along_step_update_time,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id());
    CELER_LAUNCH_KERNEL(along_step_apply_fluct_eloss,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id(),
                        device_data_.fluct);
    CELER_LAUNCH_KERNEL(along_step_update_track,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ptr<MemSpace::native>(),
                        state.ptr(),
                        this->action_id());
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
