#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2022-2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/global/alongstep/AlongStepUniformMscAction.cu
//---------------------------------------------------------------------------//
#include "AlongStepUniformMscAction.hh"

#include "corecel/device_runtime_api.h"
#include "corecel/Assert.hh"
#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/em/data/UrbanMscData.hh"
#include "celeritas/em/msc/UrbanMsc.hh"
#include "celeritas/field/DormandPrinceStepper.hh"
#include "celeritas/field/FieldDriverOptions.hh"
#include "celeritas/field/MakeMagFieldPropagator.hh"
#include "celeritas/field/UniformField.hh"
#include "celeritas/global/TrackLauncher.hh"

#include "detail/AlongStepImpl.hh"
#include "detail/MeanELoss.hh"

namespace celeritas
{
namespace
{
//---------------------------------------------------------------------------//
__global__ void
along_step_apply_msc_step_limit_kernel(DeviceCRef<CoreParamsData> const params,
                                       DeviceRef<CoreStateData> const state,
                                       DeviceCRef<UrbanMscData> const msc_data)
{
    auto launch
        = make_active_track_launcher(params,
                                     state,
                                     detail::apply_msc_step_limit<UrbanMsc>,
                                     UrbanMsc{msc_data});
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
__global__ void along_step_apply_uniform_propagation_kernel(
    DeviceCRef<CoreParamsData> const params,
    DeviceRef<CoreStateData> const state,
    UniformFieldParams const field)
{
    auto launch = make_active_track_launcher(
        params,
        state,
        detail::ApplyPropagation{},
        [&field](ParticleTrackView const& particle, GeoTrackView* geo) {
            return make_mag_field_propagator<DormandPrinceStepper>(
                UniformField(field.field), field.options, particle, geo);
        });
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
__global__ void
along_step_apply_msc_kernel(DeviceCRef<CoreParamsData> const params,
                            DeviceRef<CoreStateData> const state,
                            DeviceCRef<UrbanMscData> const msc_data)
{
    auto launch = make_active_track_launcher(
        params, state, detail::apply_msc<UrbanMsc>, UrbanMsc{msc_data});
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
__global__ void
along_step_update_time_kernel(DeviceCRef<CoreParamsData> const params,
                              DeviceRef<CoreStateData> const state)
{
    auto launch
        = make_active_track_launcher(params, state, detail::update_time);
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
__global__ void
along_step_apply_mean_eloss_kernel(DeviceCRef<CoreParamsData> const params,
                                   DeviceRef<CoreStateData> const state)
{
    using detail::MeanELoss;

    auto launch = make_active_track_launcher(
        params, state, detail::apply_eloss<MeanELoss>, MeanELoss{});
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
__global__ void
along_step_update_track_kernel(DeviceCRef<CoreParamsData> const params,
                               DeviceRef<CoreStateData> const state)
{
    auto launch
        = make_active_track_launcher(params, state, detail::update_track);
    launch(KernelParamCalculator::thread_id());
}

//---------------------------------------------------------------------------//
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Launch the along-step action on device.
 */
void AlongStepUniformMscAction::execute(CoreParams const& params,
                                        CoreStateDevice& state) const
{
    CELER_LAUNCH_KERNEL(along_step_apply_msc_step_limit,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ref<MemSpace::native>(),
                        state.ref(),
                        device_data_.msc);
    CELER_LAUNCH_KERNEL(along_step_apply_uniform_propagation,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ref<MemSpace::native>(),
                        state.ref(),
                        field_params_);
    CELER_LAUNCH_KERNEL(along_step_apply_msc,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ref<MemSpace::native>(),
                        state.ref(),
                        device_data_.msc);
    CELER_LAUNCH_KERNEL(along_step_update_time,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ref<MemSpace::native>(),
                        state);
    CELER_LAUNCH_KERNEL(along_step_apply_mean_eloss,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ref<MemSpace::native>(),
                        state);
    CELER_LAUNCH_KERNEL(along_step_update_track,
                        celeritas::device().default_block_size(),
                        state.size(),
                        params.ref<MemSpace::native>(),
                        state);
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
