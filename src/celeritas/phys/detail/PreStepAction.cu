//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/phys/detail/PreStepAction.cu
//---------------------------------------------------------------------------//
#include "PreStepAction.hh"

#include <corecel/sys/ScopedProfiling.hh>

#include "celeritas/global/ActionLauncher.device.hh"
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"
#include "celeritas/global/TrackExecutor.hh"

#include "PreStepExecutor.hh"

namespace celeritas
{
namespace detail
{
//---------------------------------------------------------------------------//
/*!
 * Launch the pre-step action on device.
 */
void PreStepAction::execute(CoreParams const& params,
                            CoreStateDevice& state) const
{
    ScopedProfiling profile_this{label()};
    TrackExecutor execute{
        params.ptr<MemSpace::native>(), state.ptr(), PreStepExecutor{}};

    static ActionLauncher<decltype(execute)> const launch_kernel(*this);
    launch_kernel(state, execute);
}

//---------------------------------------------------------------------------//
}  // namespace detail
}  // namespace celeritas
