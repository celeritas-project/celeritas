//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/phys/detail/PreStepAction.cu
//---------------------------------------------------------------------------//
#include "PreStepAction.hh"

#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"
#include "celeritas/global/LaunchAction.device.hh"
#include "celeritas/global/TrackExecutor.hh"

#include "PreStepExecutor.hh"

namespace celeritas
{
namespace detail
{
//---------------------------------------------------------------------------//
/*!
 * Launch the pre-step action on device.
 */
void PreStepAction::execute(CoreParams const& params,
                            CoreStateDevice& state) const
{
    TrackExecutor execute{
        params.ptr<MemSpace::native>(), state.ptr(), PreStepExecutor{}};

    static Launcher<decltype(execute)> const launch_kernel(*this);
    launch_kernel(state, execute);
}

//---------------------------------------------------------------------------//
}  // namespace detail
}  // namespace celeritas
