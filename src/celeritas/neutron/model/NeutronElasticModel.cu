//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2024 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/neutron/model/NeutronElasticModel.cu
//---------------------------------------------------------------------------//
#include "NeutronElasticModel.hh"

#include "celeritas/global/ActionLauncher.device.hh"
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"
#include "celeritas/global/TrackExecutor.hh"
#include "celeritas/neutron/executor/NeutronElasticExecutor.hh"
#include "celeritas/phys/InteractionApplier.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//
/*!
 * Interact with device data.
 */
void NeutronElasticModel::execute(CoreParams const& params,
                                  CoreStateDevice& state) const
{
    auto execute = make_action_track_executor(
        params.ptr<MemSpace::native>(),
        state.ptr(),
        this->action_id(),
        InteractionApplier{NeutronElasticExecutor{this->device_ref()}});
    static ActionLauncher<decltype(execute)> const launch_kernel(*this);
    launch_kernel(params, state, *this, execute);
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
