//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2024 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/neutron/model/NeutronInelasticModel.cu
//---------------------------------------------------------------------------//
#include "NeutronInelasticModel.hh"

#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//
/*!
 * Interact with device data.
 */
void NeutronInelasticModel::step(CoreParams const& params,
                                 CoreStateDevice& state) const
{
    CELER_NOT_IMPLEMENTED("Neutron inelastic interaction");
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
